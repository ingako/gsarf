#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cmath>
#include <map>
#include <iomanip>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

using namespace std;

#define IS_BIT_SET(val, pos) (val & (1 << pos))

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

template <typename T>
bool allocate_memory_on_device(T **arr, string arr_name, int count) {
    size_t memory_size = count * sizeof(T);
    cout << "\nAllocating " << memory_size << " bytes for " << arr_name << " on device..." << endl;

    hipError_t err = hipMalloc((void **) arr, memory_size); // allocate global memory on the device
    if (err) {
        cout << "error allocating memory for " << arr_name << " on device: " << memory_size << " bytes" << endl;
        return false;
    } else {
        cout << "device: memory for " << arr_name << " allocated successfully." << endl;
        return true;
    }
}

void select_k_attributes(int *reservoir, int n, int k) { 
    int i;
    for (i = 0; i < k; i++) {
        reservoir[i] = i;
    }

    for (i = k; i < n; i++) { 
        int j = rand() % i; 

        if (j < k) reservoir[j] = i; 
    }
}

vector<string> split_attributes(string line, char delim) {
    vector<string> arr;
    const char *start = line.c_str();
    bool instring = false;

    for (const char* p = start; *p; p++) {
        if (*p == '"') {
            instring = !instring;     
        } else if (*p == delim && !instring) {
            arr.push_back(string(start, p-start));
            start = p + 1;
        }
    }

    arr.push_back(string(start)); // last field delimited by end of line instead of comma
    return arr;
}

vector<string> split(string str, string delim) {
    char* cstr = const_cast<char*>(str.c_str());
    char* current;
    vector<string> arr;
    current = strtok(cstr, delim.c_str());

    while (current != NULL) {
        arr.push_back(current);
        current = strtok(NULL, delim.c_str());
    }

    return arr;
}

__device__ unsigned int get_left(unsigned int index) {
    return 2 * index + 1; 
}

__device__ unsigned int get_right(unsigned int index) {
    return 2 * index + 2;
}

__global__ void tree_traversal(int *decision_trees, 
        int *data,
        int *leaf_ids,
        int *leaf_class,
        int *leaf_back,
        int *correct_counter,
        int *samples_seen_count,
        int leaf_count_per_tree,
        int attribute_count) {
    // <<<TREE_COUNT, INSTANCE_COUNT_PER_TREE>>>
    int thread_pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_pos >= blockDim.x * gridDim.x) {
        return;
    }

    int tree_data_start_idx = blockIdx.x * blockDim.x * (attribute_count + 1); // tree_idx * instance_count_per_tree * data_row_len
    int instance_data_start_idx = threadIdx.x * (attribute_count + 1) + tree_data_start_idx;

    int *cur_data_line = data + instance_data_start_idx; 

    int pos = 0;
    while (!IS_BIT_SET(decision_trees[pos], 31)) {
        int attribute_id = decision_trees[pos];
        pos = cur_data_line[attribute_id] == 0 ? get_left(pos) : get_right(pos);
    }

    int leaf_id = (decision_trees[pos] & (~(1 << 31)));
    leaf_ids[thread_pos] = leaf_id; 
    leaf_class[thread_pos] = leaf_class[leaf_id];
    leaf_back[thread_pos] = pos;

    // printf("class code: %i  actual_class: %i   class_idx: %i\n", leaf_class_code,
    //        cur_data_line[attribute_count], data_start_idx + attribute_count);

    atomicAdd(&samples_seen_count[leaf_id + blockIdx.x * leaf_count_per_tree], 1);

    // TODO test parallel reduction
    if (leaf_class[thread_pos] == cur_data_line[attribute_count]) {
        atomicAdd(correct_counter, 1);
    }
}

__global__ void counter_increase(int *leaf_counters, 
        int *leaf_ids,
        int *data,
        int class_count,
        int attribute_count) {
    // gridDim: dim3(TREE_COUNT, INSTANCE_COUNT_PER_TREE)
    // blockDim: ATTRIBUTE_COUNT_PER_TREE
    // increment both n_ij (at row 0) and n_ijk (at row k)

    // input: an array of leaf_ids (offset) and leaf_classes built from tree_traversal

    // Each leaf counter is represented by a block and uses one thread for each attribute i and
    // value j (i.e. one thread per column) 
    //
    // Row 0 stores the total number of times value n_ij appeared.
    // Row 1 is a mask that keeps track of which attributes have been already used in internal nodes
    // along the path.
    // Row 2 and onwards stores partial counters n_ijk for each class k.

    // int block_id = blockIdx.y + blockIdx.x * gridDim.x; // instance_idx

    int block_id = blockIdx.y + blockIdx.x * gridDim.y;

    int thread_pos = threadIdx.x + block_id * blockDim.x;
    if (thread_pos >= gridDim.x * gridDim.y * blockDim.x) {
        return;
    }

    int leaf_id = leaf_ids[block_id];
    int *cur_data = data + block_id * (attribute_count + 1);

    // the counter start position corresponds to the leaf_id i.e. leaf offset
    int counter_start_pos = leaf_id * attribute_count * 2 * (class_count + 2); // TODO TREE_COUNT
    int *cur_leaf_counter = leaf_counters + counter_start_pos;
    // printf("leaf counter start pos is:  %i\n", counter_start_pos);

    int ij = cur_data[threadIdx.x] + threadIdx.x * 2; // binary value 0 or 1
    int k = cur_data[attribute_count];

    int mask = cur_leaf_counter[attribute_count * 2 + ij];
    int n_ijk_idx = (k + 2) * attribute_count * 2 + ij;

    atomicAdd(&cur_leaf_counter[ij], mask); // row 0
    atomicAdd(&cur_leaf_counter[n_ijk_idx], mask);
}

__global__ void compute_information_gain(int *leaf_counters, 
        float *info_gain_vals, 
        int class_count) {
    // each leaf_counter is mapped to one block in the 1D grid
    // each block needs as many threads as twice number of the (binary) attributes

    // output: a vector with the attributes information gain  values for all leaves in each of the trees
    // gridDim: dim3(TREE_COUNT, LEAF_COUNT_PER_TREE)
    // blockDim: attributes_per_tree * 2 (equal to a info_gain_vals per leaf)

    int thread_pos = threadIdx.x + gridDim.x * blockIdx.x;
    if (thread_pos >= gridDim.x * blockDim.x) {
        return;
    }

    int tree_id = blockIdx.x;
    int leaf_id = blockIdx.y;

    int leaf_count_per_tree = gridDim.y;
    int leaf_counter_row_len = blockDim.x;
    int leaf_counter_size = blockDim.x * (class_count + 2);

    int cur_tree_start_pos = tree_id * leaf_count_per_tree * leaf_counter_size;
    int cur_leaf_start_pos = cur_tree_start_pos + leaf_id * leaf_counter_size;
    int *cur_leaf_counter = leaf_counters + cur_leaf_start_pos;

    int cur_tree_info_gain_start_pos = tree_id * leaf_count_per_tree * leaf_counter_row_len;
    int cur_leaf_info_gain_start_pos = cur_tree_info_gain_start_pos + leaf_id *
        leaf_counter_row_len;
    float *cur_info_gain_vals = info_gain_vals + cur_leaf_info_gain_start_pos; // TODO


    int mask = 0;
    int a_ij = cur_leaf_counter[threadIdx.x];
    float sum = 0.0;

    // sum up a column
    for (int i = 0; i < class_count; i++) {
        int a_ijk = cur_leaf_counter[threadIdx.x + (2 + i) * leaf_counter_row_len];

        // float param = a_ijk / a_ij; // TODO float division by zero returns INF
        // asm("max.f32 %0, %1, %2;" : "=f"(param) : "f"(param), "f"((float) 0.0));
        // sum += -(param) * log(param);

        float param = 0.0;
        if (a_ijk != 0 && a_ij != 0) {
            param = (float) a_ijk / a_ij;
        }

        float log_param = 0.0;
        if (abs(param) > 0.00001) {
            log_param = log(param);
        }

        sum += -(param) * log_param;
    }

    cur_info_gain_vals[threadIdx.x] = -sum;

    __syncthreads();

    float i_00 = 0, i_01 = 0;
    int i_idx = 0;

    if (threadIdx.x % 2 == 0) {
        i_00 = cur_info_gain_vals[threadIdx.x];
        i_01 = cur_info_gain_vals[threadIdx.x + 1];
        i_idx = (threadIdx.x >> 1);
    }

    __syncthreads();

    if (threadIdx.x % 2 == 0) {
        cur_info_gain_vals[i_idx] = i_00 + i_01;
    }
}

// hoeffding bound
// providing an upper bound on the probability that the sum of a sample of independent random
// variables deviates from its expected value
// 
// range: range of the random variable
// confidence: desired probability of the estimate not being within the expected value
// n: the number of examples collected at the node
__device__ float compute_hoeffding_bound(float range, float confidence, float n) {
    return sqrt(((range * range) * log(1.0 / confidence)) / (2.0 * n));
}

__global__ void node_split(float *info_gain_vals, 
        int *attribute_idx_arr,
        unsigned int *node_split_decisions, 
        int attribute_count,
        float r,
        float delta,
        int *samples_seen_count) {
    // <<<TREE_COUNT, LEAF_COUNT_PER_TREE>>>
    // note: different from paper by using one thread per leaf
    // output: an array of decisions 
    //         - the most significant bit denotes whether a leaf needs to be split
    //         - the rest bits denote the attribute id to split on

    int thread_pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_pos >= gridDim.x * blockDim.x) {
        return;
    }

    int tree_idx = blockIdx.x;
    int leaf_idx = threadIdx.x;
    int leaf_count_per_tree = blockDim.x;

    int cur_tree_start_pos = tree_idx * leaf_count_per_tree * attribute_count;
    int cur_leaf_start_pos = cur_tree_start_pos + leaf_idx * attribute_count;
    int *cur_attribute_idx_arr = attribute_idx_arr + cur_leaf_start_pos;

    int cur_tree_info_gain_start_pos = tree_idx * leaf_count_per_tree * attribute_count * 2;
    int cur_leaf_info_gain_start_pos = cur_tree_info_gain_start_pos + leaf_idx * 
        attribute_count * 2;
    float *cur_info_gain_vals = info_gain_vals + cur_leaf_start_pos;

    thrust::sort_by_key(thrust::seq, 
            cur_info_gain_vals, 
            cur_info_gain_vals + attribute_count,
            cur_attribute_idx_arr);

    float first_best = cur_info_gain_vals[attribute_count - 1];
    float second_best = cur_info_gain_vals[attribute_count - 2];

    float hoeffding_bound = compute_hoeffding_bound(r, delta, samples_seen_count[thread_pos]);

    unsigned int decision = 0;
    if (first_best - second_best > hoeffding_bound) {
        // split on the best attribute
        decision |= (1 << 31);
        decision |= cur_attribute_idx_arr[attribute_count - 1];
    }

    node_split_decisions[thread_pos] = decision;
}

int main(void) {
    const int TREE_COUNT = 1;
    cout << "Number of decision trees: " << TREE_COUNT << endl;

    const int INSTANCE_COUNT_PER_TREE = 10;
    cout << "Instance count per tree: " << INSTANCE_COUNT_PER_TREE << endl;

    // hoeffding bound parameters
    float n_min = TREE_COUNT * INSTANCE_COUNT_PER_TREE;
    float delta = 0.05; // pow((float) 10.0, -7);
    float r = 1;
    cout << "hoeffding bound parameters: " << endl
        << "n_min: " << n_min << endl
        << "delta: " << delta << endl
        << "r    : " << r     << endl;

    // Use a different seed value for each run
    // srand(time(NULL));

    cout << "Reading class file..." << endl; 
    ifstream class_file("data/random-tree/labels.txt");
    string class_line;

    // init mapping between class and code
    map<string, int> class_code_map;
    map<int, string> code_class_map;

    vector<string> class_arr = split(class_line, " ");
    string code_str, class_str;

    int line_count = 0;
    while (class_file >> code_str >> class_str) {
        int code_int = atoi(code_str.c_str());
        class_code_map[class_str] = code_int;
        code_class_map[code_int] = class_str;
        line_count++;
    }
    const int CLASS_COUNT = line_count; 
    cout << "Number of classes: " << CLASS_COUNT << endl;

    // prepare attributes
    std::ifstream file("data/random-tree/synthetic_with_noise.csv");
    string line;

    getline(file, line);
    // const int ATTRIBUTE_COUNT_TOTAL = split_attributes(line, ',').size() - 2; // for activity-recognition dataset
    const int ATTRIBUTE_COUNT_TOTAL = split(line, ",").size() - 1;
    const int ATTRIBUTE_COUNT_PER_TREE = (int) sqrt(ATTRIBUTE_COUNT_TOTAL);

    cout << "Attribute count total: " << ATTRIBUTE_COUNT_TOTAL << endl;
    cout << "Attribute count per tree: " << ATTRIBUTE_COUNT_PER_TREE << endl;

    const unsigned int NODE_COUNT_PER_TREE = (1 << (ATTRIBUTE_COUNT_PER_TREE + 1));
    const unsigned int LEAF_COUNT_PER_TREE = (1 << ATTRIBUTE_COUNT_PER_TREE);

    cout << "NODE_COUNT_PER_TREE: " << NODE_COUNT_PER_TREE << endl;
    cout << "LEAF_COUNT_PER_TREE: " << LEAF_COUNT_PER_TREE << endl;

    size_t memory_size;

    // select k random attributes for each tree
    int attribute_arr[TREE_COUNT * ATTRIBUTE_COUNT_PER_TREE];
    for (int i = 0; i < TREE_COUNT; i++) {
        select_k_attributes(attribute_arr + i * ATTRIBUTE_COUNT_PER_TREE, 
                ATTRIBUTE_COUNT_TOTAL, ATTRIBUTE_COUNT_PER_TREE);
    }

    cout << "\nAttributes selected per tree: " << endl;
    for (int i = 0; i < TREE_COUNT; i++) {
        cout << "tree " << i << endl;
        for (int j = 0; j < ATTRIBUTE_COUNT_PER_TREE; j++) {
            cout << attribute_arr[i * ATTRIBUTE_COUNT_PER_TREE + j] << " ";
        }
        cout << endl;
    }

    // init decision tree
    cout << "\nAllocating memory on host..." << endl;
    // void *allocated = malloc(NODE_COUNT_PER_TREE * TREE_COUNT * sizeof(int));
    void *allocated = calloc(NODE_COUNT_PER_TREE * TREE_COUNT, sizeof(int)); // TODO
    if (allocated == NULL) {
        cout << "host error: memory allocation for decision trees failed" << endl;
        return 1;
    }
    int *h_decision_trees = (int*) allocated;
    int *d_decision_trees;

    allocated = malloc(LEAF_COUNT_PER_TREE * TREE_COUNT * sizeof(int));
    if (allocated == NULL) {
        cout << "host error: memory allocation for leaf_class failed" << endl;
        return 1;
    }
    int *h_leaf_class = (int*) allocated; // stores the class for a given leaf
    int *d_leaf_class;

    allocated = malloc(LEAF_COUNT_PER_TREE * TREE_COUNT * sizeof(int));
    if (allocated == NULL) {
        cout << "host error: memory allocation for leaf_back failed" << endl;
        return 1;
    }
    int *h_leaf_back = (int*) allocated; // reverse pointer to map a leaf id to an offset in the tree array
    int *d_leaf_back;

    cout << "Init: set root as leaf for each tree in the forest..." << endl;
    for (int i = 0; i < TREE_COUNT; i++) {
        h_decision_trees[i * NODE_COUNT_PER_TREE] = (1 << 31); // init root node
    }

    cout << "\nAllocating memory on device..." << endl;

    if (!allocate_memory_on_device(&d_decision_trees, "decision_trees", NODE_COUNT_PER_TREE * TREE_COUNT)) {
        return 1;
    }
    gpuErrchk(hipMemcpy(d_decision_trees, h_decision_trees, NODE_COUNT_PER_TREE * TREE_COUNT 
                * sizeof(int), hipMemcpyHostToDevice));

    int *d_leaf_ids;
    if (!allocate_memory_on_device(&d_leaf_ids, "leaf_ids", LEAF_COUNT_PER_TREE * TREE_COUNT)) {
        return 1;
    }

    if (!allocate_memory_on_device(&d_leaf_class, "leaf_class", LEAF_COUNT_PER_TREE * TREE_COUNT)) {
        return 1;
    }

    if (!allocate_memory_on_device(&d_leaf_back, "leaf_back", LEAF_COUNT_PER_TREE * TREE_COUNT)) {
        return 1;
    }

    // TODO: for testing only
    int leaf_counter_size = ATTRIBUTE_COUNT_PER_TREE * 2 * (CLASS_COUNT + 2);
    int all_leaf_counters_size = TREE_COUNT * LEAF_COUNT_PER_TREE * leaf_counter_size;

    // int *h_leaf_counters = (int*) malloc(all_leaf_counters_size * sizeof(int));
    int *h_leaf_counters = (int*) calloc(all_leaf_counters_size, sizeof(int));

    // init mask row
    for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
        int *cur_tree_leaf_counters = h_leaf_counters + tree_idx * LEAF_COUNT_PER_TREE *
            leaf_counter_size;
        for (int leaf_idx = 0; leaf_idx < LEAF_COUNT_PER_TREE; leaf_idx++) {
            int *cur_leaf_counter = cur_tree_leaf_counters + leaf_idx * leaf_counter_size;
            int *cur_leaf_counter_mask_row = cur_leaf_counter + ATTRIBUTE_COUNT_PER_TREE * 2;

            for (int k = 0; k < ATTRIBUTE_COUNT_PER_TREE * 2; k++) {
                cur_leaf_counter_mask_row[k] = 1;
            }
        }
    }

    int *d_leaf_counters;
    if (!allocate_memory_on_device(&d_leaf_counters, "leaf_counters", all_leaf_counters_size)) {
        return 1;
    }
    gpuErrchk(hipMemcpy(d_leaf_counters, h_leaf_counters, all_leaf_counters_size * sizeof(int),
                hipMemcpyHostToDevice));

    // TODO: h_info_gain_vals for testing only
    int info_gain_vals_len = TREE_COUNT * LEAF_COUNT_PER_TREE * ATTRIBUTE_COUNT_PER_TREE * 2;
    float *h_info_gain_vals = (float*) malloc(info_gain_vals_len * sizeof(float));

    float *d_info_gain_vals;
    if (!allocate_memory_on_device(&d_info_gain_vals, "info_gain_vals", info_gain_vals_len)) {
        return 1;
    }

    // allocate memory for attribute indices on host for computing information gain
    int *h_attribute_idx_arr;
    int *d_attribute_idx_arr;
    int attribute_idx_arr_len = TREE_COUNT * LEAF_COUNT_PER_TREE * ATTRIBUTE_COUNT_PER_TREE; 
    
    allocated = malloc(attribute_idx_arr_len * sizeof(int));
    if (allocated == NULL) {
        cout << "host error: memory allocation for h_attribute_idx_arr failed" << endl;
        return 1;
    }
    h_attribute_idx_arr = (int*) allocated;

    if (!allocate_memory_on_device(&d_attribute_idx_arr, "attribute_idx_arr",
                attribute_idx_arr_len)) {
        return 1;
    }

    for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
        int *cur_tree_attribute_idx_arr = h_attribute_idx_arr + tree_idx * LEAF_COUNT_PER_TREE
            * ATTRIBUTE_COUNT_PER_TREE;

        for (int leaf_idx = 0; leaf_idx < LEAF_COUNT_PER_TREE; leaf_idx++) {
            int *cur_attribute_idx_arr = cur_tree_attribute_idx_arr + leaf_idx *
                ATTRIBUTE_COUNT_PER_TREE;
            for (int i = 0; i < ATTRIBUTE_COUNT_PER_TREE; i++) {
                cur_attribute_idx_arr[i] = i;
            }
        }
    }

    // allocate memory for node_split_decisions
    unsigned int *h_node_split_decisions;
    unsigned int *d_node_split_decisions;
    int node_split_decisions_len = LEAF_COUNT_PER_TREE * TREE_COUNT;

    allocated = malloc(node_split_decisions_len * sizeof(unsigned int));
    if (allocated == NULL) {
        cout << "host error: memory allocation for h_node_split_decisions failed" << endl;
        return 1;
    }
    h_node_split_decisions = (unsigned int*) allocated;

    if (!allocate_memory_on_device(&d_node_split_decisions, "node_split_decisions", 
                node_split_decisions_len)) {
        return 1;
    }
    
    int samples_seen_count_len = TREE_COUNT * LEAF_COUNT_PER_TREE;
    int *h_samples_seen_count = (int*) malloc(samples_seen_count_len * sizeof(int));
    int *d_samples_seen_count;
    if (!allocate_memory_on_device(&d_samples_seen_count, "samples_seen_count",
                samples_seen_count_len)) {
        return 1;
    }
    gpuErrchk(hipMemcpy((void *) d_samples_seen_count, (void *) h_samples_seen_count, 
                samples_seen_count_len * sizeof(int), hipMemcpyHostToDevice));


    cout << "\nInitializing training data arrays..." << endl;

    int data_len = TREE_COUNT * INSTANCE_COUNT_PER_TREE * (ATTRIBUTE_COUNT_PER_TREE + 1);
    int *h_data = (int*) malloc(data_len * sizeof(int));

    int *d_data;
    if (!allocate_memory_on_device(&d_data, "data", data_len)) {
        return 1;
    }

    vector<string> raw_data_row;
    int data_start_idx;

    int block_count;
    int thread_count;

    int cur_node_count_per_tree[TREE_COUNT] = { 1 }; // initialize all counts to 1
    int cur_leaf_count_per_tree[TREE_COUNT] = { 1 };
    
    cout << endl << "=====Training Start=====" << endl;

    int h_correct_counter = 0;
    int *d_correct_counter;
    hipMalloc((void **) &d_correct_counter, sizeof(int));

    bool eof = false;

    while (!eof) {

        cout << "preparing data..." << endl;
        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
            int *cur_attribute_arr = attribute_arr + tree_idx * ATTRIBUTE_COUNT_PER_TREE;
            int *cur_tree_data_rows = h_data + (ATTRIBUTE_COUNT_PER_TREE + 1) * INSTANCE_COUNT_PER_TREE *
                tree_idx;

            for (int instance_idx = 0; instance_idx < INSTANCE_COUNT_PER_TREE; instance_idx++) {
                if (!getline(file, line)) {
                    eof = true;
                    break;
                }

                raw_data_row = split(line, ",");

                int *cur_data_row = cur_tree_data_rows + (ATTRIBUTE_COUNT_PER_TREE + 1) * instance_idx;

                for (int i = 0; i < ATTRIBUTE_COUNT_PER_TREE; i++) {
                    int attribute_code = cur_attribute_arr[i];
                    int val = strcmp(raw_data_row[attribute_code].c_str(), (const char*) "value1") == 0 ? 0 : 1;

                    cur_data_row[i] = val;

                    // cout << "a" << attribute_arr[attribute_offset] << ":";
                    // cout << h_data[data_start_idx + i] << " ";
                }
                // cout << endl;

                cur_data_row[ATTRIBUTE_COUNT_PER_TREE] = class_code_map[raw_data_row[raw_data_row.size() - 1]]; // class
                // cout << "h_data_class_idx: " << data_start_idx + ATTRIBUTE_COUNT_PER_TREE << endl;
                // cout << "h_data class: " << h_data[data_start_idx + ATTRIBUTE_COUNT_PER_TREE] << endl;
            }

            if (eof) {
                break; // TODO
            }
        }
        if (eof) break; // TODO

        gpuErrchk(hipMemcpy((void *) d_data, (void *) h_data, data_len * sizeof(int), hipMemcpyHostToDevice));

        // TODO to remove
        gpuErrchk(hipMemcpy(d_decision_trees, h_decision_trees, NODE_COUNT_PER_TREE * TREE_COUNT 
                    * sizeof(int), hipMemcpyHostToDevice));

        cout << "\nlaunching tree_traversal kernel..." << endl;

        block_count = TREE_COUNT;
        thread_count = INSTANCE_COUNT_PER_TREE;

        hipMemset(d_correct_counter, 0, sizeof(int));

        cout << "launching " << block_count * thread_count << " threads for tree_traversal" << endl;

        tree_traversal<<<block_count, thread_count>>>(d_decision_trees,
                d_data,
                d_leaf_ids,
                d_leaf_class,
                d_leaf_back,
                d_correct_counter,
                d_samples_seen_count,
                LEAF_COUNT_PER_TREE,
                ATTRIBUTE_COUNT_PER_TREE);

        hipDeviceSynchronize();
        hipGetLastError();

        gpuErrchk(hipMemcpy(h_decision_trees, d_decision_trees, TREE_COUNT * NODE_COUNT_PER_TREE *
                    sizeof(int), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy((void *) h_samples_seen_count, (void *) d_samples_seen_count, samples_seen_count_len *
                    sizeof(int), hipMemcpyDeviceToHost));

        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
            cout << "tree " << tree_idx << endl;
            int *cur_decision_tree = h_decision_trees + tree_idx * NODE_COUNT_PER_TREE;
            int *cur_samples_seen_count = h_samples_seen_count + tree_idx * LEAF_COUNT_PER_TREE;

            for (int i = 0; i < NODE_COUNT_PER_TREE; i++) {
                cout << cur_decision_tree[i] << " ";
            }
            cout << endl;

            cout << "samples seen count: " << endl;
            for (int i = 0; i < LEAF_COUNT_PER_TREE; i++) {
                cout << cur_samples_seen_count[i] << " ";
            }
            cout << endl;
        }

        cout << "tree_traversal completed" << endl;

        gpuErrchk(hipMemcpy(&h_correct_counter, d_correct_counter, sizeof(int), hipMemcpyDeviceToHost));
        cout << "h_correct_counter: " << h_correct_counter << endl;
        double accuracy = (double) h_correct_counter / (INSTANCE_COUNT_PER_TREE * TREE_COUNT);
        cout << INSTANCE_COUNT_PER_TREE * TREE_COUNT << ": " << accuracy << endl;

        cout << "\nlaunching counter_increase kernel..." << endl;

        counter_increase
            <<<dim3(TREE_COUNT, INSTANCE_COUNT_PER_TREE), ATTRIBUTE_COUNT_PER_TREE>>>(
                    d_leaf_counters,
                    d_leaf_ids,
                    d_data,
                    CLASS_COUNT,
                    ATTRIBUTE_COUNT_PER_TREE);

        hipDeviceSynchronize();

        // TODO: for testing only
        gpuErrchk(hipMemcpy(h_leaf_counters, d_leaf_counters, all_leaf_counters_size 
                    * sizeof(int), hipMemcpyDeviceToHost));

        cout << "counter_increase result: " << endl;

        int row_len = ATTRIBUTE_COUNT_PER_TREE * 2;
        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
            cout << "tree " << tree_idx << endl;

            int *cur_tree_leaf_counter = h_leaf_counters + tree_idx * LEAF_COUNT_PER_TREE
                * leaf_counter_size;

            for (int leaf_idx = 0; leaf_idx < cur_leaf_count_per_tree[tree_idx]; leaf_idx++) {
                int *cur_leaf_counter = cur_tree_leaf_counter + leaf_idx * leaf_counter_size;

                for (int k = 0; k < CLASS_COUNT + 2; k++) {
                    cout << "row " << k << ": ";
                    for (int ij = 0; ij < row_len; ij++) {
                        cout << right << setw(8) <<  cur_leaf_counter[k * row_len + ij] << " ";
                    }
                    cout << endl;
                }
            }
            cout << endl;
        }


        cout << "\nlanuching compute_information_gain kernel..." << endl;

        dim3 grid(TREE_COUNT, LEAF_COUNT_PER_TREE);
        thread_count = ATTRIBUTE_COUNT_PER_TREE * 2;
        compute_information_gain<<<grid, thread_count>>>(d_leaf_counters,
                d_info_gain_vals,
                CLASS_COUNT);

        // log info_gain_vals
        gpuErrchk(hipMemcpy(h_info_gain_vals, d_info_gain_vals, info_gain_vals_len *
                    sizeof(float), hipMemcpyDeviceToHost));

        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
            cout << "tree " << tree_idx << endl;
            int cur_tree_info_gain_vals_start_pos = tree_idx * LEAF_COUNT_PER_TREE *
                ATTRIBUTE_COUNT_PER_TREE * 2;

            for (int leaf_idx = 0; leaf_idx < LEAF_COUNT_PER_TREE; leaf_idx++) {
                int cur_info_gain_vals_start_pos = cur_tree_info_gain_vals_start_pos + leaf_idx *
                    ATTRIBUTE_COUNT_PER_TREE * 2;
                float *cur_info_gain_vals = h_info_gain_vals + cur_info_gain_vals_start_pos;

                for (int i = 0; i < ATTRIBUTE_COUNT_PER_TREE; i++) {
                    cout << cur_info_gain_vals[i] << " ";
                }
                cout << endl;
            }
            cout << endl;
        }

        cout << "compute_information_gain completed" << endl;

        cout << "\nlaunching node_split kernel..." << endl;

        gpuErrchk(hipMemcpy(d_attribute_idx_arr, h_attribute_idx_arr, attribute_idx_arr_len *
                    sizeof(int), hipMemcpyHostToDevice));

        node_split<<<TREE_COUNT, LEAF_COUNT_PER_TREE>>>(d_info_gain_vals,
                d_attribute_idx_arr,
                d_node_split_decisions,
                ATTRIBUTE_COUNT_PER_TREE,
                r,
                delta,
                d_samples_seen_count);

        gpuErrchk(hipMemcpy(h_node_split_decisions, d_node_split_decisions,
                    node_split_decisions_len * sizeof(int), hipMemcpyDeviceToHost));

        gpuErrchk(hipMemcpy(h_leaf_back, d_leaf_back, TREE_COUNT * LEAF_COUNT_PER_TREE *
                    sizeof(int), hipMemcpyDeviceToHost));

        // TODO perform actual node split on device
        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
            cout << "tree " << tree_idx << endl; 

            int tree_leaf_start_pos = tree_idx * LEAF_COUNT_PER_TREE;
            unsigned int *cur_node_split_decisions = h_node_split_decisions + tree_leaf_start_pos;

            int *cur_decision_tree = h_decision_trees + tree_idx * NODE_COUNT_PER_TREE;
            int *cur_leaf_counters = h_leaf_counters + tree_idx * leaf_counter_size;

            int cur_node_count = cur_node_count_per_tree[tree_idx];
            if (cur_node_count_per_tree[tree_idx] == NODE_COUNT_PER_TREE) {
                continue;
            }

            int tree_counter_start_pos = tree_idx * leaf_counter_size * LEAF_COUNT_PER_TREE;
            int *cur_tree_leaf_counters = h_leaf_counters + tree_counter_start_pos;

            for (int leaf_idx = 0; leaf_idx < LEAF_COUNT_PER_TREE; leaf_idx++) {
                int *cur_leaf_counter = cur_tree_leaf_counters + leaf_idx * LEAF_COUNT_PER_TREE;

                unsigned int decision = cur_node_split_decisions[leaf_idx];
                cout << decision << " ";

                if (!IS_BIT_SET(decision, 31)) {
                    continue;
                }

                h_samples_seen_count[leaf_idx + tree_leaf_start_pos] = 0;

                int attribute_id = (decision & ~(1 << 31));
                cur_decision_tree[h_leaf_back[leaf_idx + tree_leaf_start_pos]] = attribute_id;

                int class0_count = cur_leaf_counter[ATTRIBUTE_COUNT_PER_TREE * 2 * 2 + attribute_id
                    * 2];
                int class1_count = cur_leaf_counter[ATTRIBUTE_COUNT_PER_TREE * 2 * 3 + attribute_id
                    * 2];

                int left_class = 0, right_class = 1;
                if (class0_count < class1_count) {
                    left_class = 1;
                    right_class = 0;
                }

                cur_decision_tree[cur_node_count] = (1 << 31) | left_class;
                cur_decision_tree[cur_node_count + 1] = (1 << 31) | right_class;

                cur_node_count_per_tree[tree_idx] += 2;

                // reset current leaf_counter and add copy mask to a new leaf counter
                int *new_leaf_counter = cur_tree_leaf_counters + cur_leaf_count_per_tree[tree_idx] *
                    leaf_counter_size;

                cur_leaf_count_per_tree[tree_idx] += 1;

                for (int k = 0; k < CLASS_COUNT * 2; k++) {
                    int *cur_leaf_counter_row = cur_leaf_counter + ATTRIBUTE_COUNT_PER_TREE * 2 * k;
                    int *new_leaf_counter_row = new_leaf_counter + ATTRIBUTE_COUNT_PER_TREE * 2 * k;

                    if (k == 1) {
                        for (int ij = 0; ij < ATTRIBUTE_COUNT_PER_TREE * 2; ij++) {
                            if (ij == attribute_id * 2 || ij == attribute_id * 2 + 1) {
                                cur_leaf_counter_row[ij] = 0;

                            } else {
                                // copy to new leaf
                                new_leaf_counter_row[ij] = 1;
                            }
                        }
                        continue;
                    }

                    for (int ij = 0; ij < ATTRIBUTE_COUNT_PER_TREE * 2; ij++) {
                        cur_leaf_counter_row[ij] = 0;
                    }
                }
            }
            cout << endl;
        }

        cout << "node_split completed" << endl;

        break; // TODO
    }

    hipFree(d_decision_trees);
    hipFree(d_leaf_ids);
    hipFree(d_leaf_class);
    hipFree(d_leaf_back);
    hipFree(d_leaf_counters);
    hipFree(d_data);
    hipFree(d_info_gain_vals);
    hipFree(d_node_split_decisions);

    return 0;
}
