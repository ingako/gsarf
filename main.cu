
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cmath>
#include <map>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void select_k_attributes(int *reservoir, int n, int k) { 
    int i;
    for (i = 0; i < k; i++) {
        reservoir[i] = i;
    }

    for (i = k; i < n; i++) { 
        int j = rand() % i; 

        if (j < k) reservoir[j] = i; 
    }
}

vector<string> split_attributes(string line, char delim) {
    vector<string> arr;
    const char *start = line.c_str();
    bool instring = false;

    for (const char* p = start; *p; p++) {
        if (*p == '"') {
            instring = !instring;     
        } else if (*p == delim && !instring) {
            arr.push_back(string(start, p-start));
            start = p + 1;
        }
    }

    arr.push_back(string(start)); // last field delimited by end of line instead of comma
    return arr;
}

vector<string> split(string str, string delim) {
    char* cstr = const_cast<char*>(str.c_str());
    char* current;
    vector<string> arr;
    current = strtok(cstr, delim.c_str());

    while (current != NULL) {
        arr.push_back(current);
        current = strtok(NULL, delim.c_str());
    }

    return arr;
}

__device__ bool is_leaf(unsigned int node) {
    return ((node >> 31) & 1) == 1;
}

__device__ unsigned int get_left(unsigned int index) {
    return 2 * index + 1; 
}

__device__ unsigned int get_right(unsigned int index) {
    return 2 * index + 2;
}

__global__ void tree_traversal(int *decision_trees, 
        int *attribute_arr, 
        int *data,
        int *leaf_class,
        int *leaf_back,
        int attribute_count) {
    int pos = 0;
    int thread_pos = threadIdx.x + blockIdx.x * blockDim.x;
    int data_start_pos = data[thread_pos];

    while (!is_leaf(decision_trees[pos])) {
        for (int i = 0; i < attribute_count; i++) {
            if (attribute_arr[i] != decision_trees[pos]) {
                continue;
            }
            pos = data[data_start_pos + i] < 0 ? get_left(pos) : get_right(pos);
        }
    }

    leaf_class[thread_pos] = (decision_trees[pos] & (~(1 << 31)));
    leaf_back[thread_pos] = pos;
}

__global__ void counter_increase(int *leaf_counters) {
    // TODO
}

__global__ void compute_information_gain(int *leaf_counters, 
        int *info_gain_vals, 
        int class_count) {
    // each leaf_counter is mapped to one block in the 1D grid
    // each block needs as many threads as twice number of the (binary) attributes
    // output: a vector with the attributes information gain  values for all leaves in each of the trees

    // gridDim: dim3(forest_size, leaf_count)
    // blockDim: attributes_per_tree * 2

    int tree_id = blockIdx.x;
    int tree_count = gridDim.x;
    int leaf_id = blockIdx.y;
    int leaf_count = gridDim.y;

    int block_id = blockIdx.x + blockIdx.y * gridDim.x; 
    int thread_pos = threadIdx.x + block_id * blockDim.x;

    int *cur_leaf_counter_col = leaf_counters + thread_pos; // TODO
    
    int a_ij = cur_leaf_counter_col[0];
    int sum = 0;


    for (int i = 0; i < class_count; i++) {
        int a_ijk = cur_leaf_counter_col[2 + i];
        
        float param = a_ijk / a_ij; // TODO float division by zero returns INF
        asm("max.f32 %0, %1, %2;" : "=f"(param) : "f"(param), "f"((float) 0.0));
        sum += -(param) * log(param);
    }
    
    info_gain_vals[thread_pos] = -sum;

    __syncthreads();
    
    int i_00 = 0, i_01 = 0, i_idx = 0;

    if (threadIdx.x % 2 == 0) {
        i_00 = info_gain_vals[thread_pos];
        i_01 = info_gain_vals[thread_pos + 1];
        i_idx = (threadIdx.x << 1) + block_id * blockDim.x;
    }

    __syncthreads();

    if (threadIdx.x % 2 == 0) {
        info_gain_vals[i_idx] = i_00 + i_01;
    }
}

int main(void) {
    const int FOREST_SIZE = 1;
    cout << "Forest size: " << FOREST_SIZE << endl;

    const int INSTANCE_COUNT_PER_TREE = 1;
    cout << "Instance count per tree: " << INSTANCE_COUNT_PER_TREE << endl;
    
    // Use a different seed value for each run
    // srand(time(NULL));
    
    cout << "Reading class file..." << endl; 
    ifstream class_file("data/activity_labels.txt");
    string class_line;

    // init mapping between class and code
    map<string, int> class_code_map;
    map<int, string> code_class_map;

    vector<string> class_arr = split(class_line, " ");
    string code_str, class_str;
    
    int line_count = 0;
    while (class_file >> code_str >> class_str) {
        int code_int = atoi(code_str.c_str());
        class_code_map[class_str] = code_int;
        code_class_map[code_int] = class_str;
        line_count++;
    }
    const int CLASS_COUNT = line_count; 
    cout << "Number of class: " << CLASS_COUNT << endl;

    // prepare attributes
    std::ifstream file("data/train.csv");
    string line;

    getline(file, line);
    const int ATTRIBUTE_COUNT_TOTAL = split_attributes(line, ',').size() - 2; 
    const int ATTRIBUTE_COUNT_PER_TREE = (int) sqrt(ATTRIBUTE_COUNT_TOTAL);

    cout << "Attribute count total: " << ATTRIBUTE_COUNT_TOTAL << endl;
    cout << "Attribute count per tree: " << ATTRIBUTE_COUNT_PER_TREE << endl;

    const unsigned int TREE_NODE_COUNT = (1 << ATTRIBUTE_COUNT_PER_TREE);
    const unsigned int LEAF_COUNT = (TREE_NODE_COUNT >> 1);

    cout << "TREE_NODE_COUNT: " << TREE_NODE_COUNT << " bytes" << endl;
    cout << "LEAF_COUNT: " << LEAF_COUNT << " bytes" << endl;

    // select k random attributes for each tree
    int h_attribute_arr[FOREST_SIZE][ATTRIBUTE_COUNT_PER_TREE];
    for (int i = 0; i < FOREST_SIZE; i++) {
        select_k_attributes(h_attribute_arr[i], ATTRIBUTE_COUNT_TOTAL, ATTRIBUTE_COUNT_PER_TREE);
    }

    // init decision tree
    void *allocated = malloc(TREE_NODE_COUNT * sizeof(int));
    if (allocated == NULL) {
        cout << "host error: memory allocation for decision trees failed" << endl;
        return 1;
    }
    int *h_decision_trees = (int*) allocated;
    int *d_decision_trees;

    allocated = malloc(LEAF_COUNT * sizeof(int));
    if (allocated == NULL) {
        cout << "host error: memory allocation for leaf_class failed" << endl;
        return 1;
    }
    int *h_leaf_class = (int*) allocated; // stores the class for a given leaf
    int *d_leaf_class;

    allocated = malloc(LEAF_COUNT * sizeof(int));
    if (allocated == NULL) {
        cout << "host error: memory allocation for leaf_back failed" << endl;
        return 1;
    }
    int *h_leaf_back = (int*) allocated; // reverse pointer to map a leaf id to an offset in the tree array
    int *d_leaf_back;

    // int h_leaf_counters[(2 + CLASS_COUNT) * ATTRIBUTE_COUNT_PER_TREE * 2 *
    //    LEAF_COUNT * FOREST_SIZE];
    int *d_leaf_counters;
    
    cout << "Init: set root as leaf for each tree in the forest..." << endl;
    for (int i = 0; i < FOREST_SIZE; i++) {
        h_decision_trees[i * TREE_NODE_COUNT] = 0;
        h_decision_trees[i * TREE_NODE_COUNT] |= (1 << 31); // init root node
    }

    cout << "Allocating memory on device..." << endl;

    hipError_t err;

    cout << "Allocating  " << TREE_NODE_COUNT * FOREST_SIZE * sizeof(int) << " bytes for decision trees on device..." << endl;
    err = hipMalloc((void **) &d_decision_trees, TREE_NODE_COUNT *
            FOREST_SIZE * sizeof(int)); // allocate global memory on the device
    if (err) {
        cout << "error allocating memory for decision_trees on device: " <<
            TREE_NODE_COUNT * FOREST_SIZE << " bytes" << endl;
        return 1;
    } else {
        cout << "device: memory for decision tree allocated successfully" <<
            endl;
    }
    
    cout << "Allocating " << LEAF_COUNT * sizeof(int) << " bytes for leaf_class on device..." << endl;
    err = hipMalloc((void **) &d_leaf_class, LEAF_COUNT * sizeof(int));
    if (err) {
        cout << "error allocating memory for leaf_class on device" << endl;
        return 1;
    } else {
        cout << "device: memory for leaf_class allocated successfully. " << endl;
    }

    cout << "Allocating " << LEAF_COUNT * sizeof(int) << " bytes for leaf_back on device..." << endl;
    err = hipMalloc((void **) &d_leaf_back, LEAF_COUNT * sizeof(int));
    if (err) {
        cout << "error allocating memory for leaf_back on device" << endl;
        return 1;
    } else {
        cout << "device: memory for leaf_back allocated successfully." << endl;
    }

    gpuErrchk(hipMemcpy(d_decision_trees, h_decision_trees, TREE_NODE_COUNT *
                FOREST_SIZE * sizeof(int), hipMemcpyHostToDevice));

    cout << "Initialize training data arrays..." << endl;
    int tree_idx = 0;
    int instance_idx = 0;

    int *h_data = (int*) malloc(INSTANCE_COUNT_PER_TREE * (ATTRIBUTE_COUNT_PER_TREE + 1) * sizeof(int));
    int *d_data;
    
    err = hipMalloc((void**) &d_data, INSTANCE_COUNT_PER_TREE * (ATTRIBUTE_COUNT_PER_TREE + 1) * sizeof(int));
    if (err) {
        cout << "error allocating memory for data array on GPU" << endl;
        return 1;
    }

    int *d_cur_attribute_arr;
    vector<string> arr;

    int block_count;
    int thread_count;

    cout << endl << "Start training..." << endl;
    while (getline(file, line)) {
        arr = split(line, ",");

        int *cur_attribute_arr = h_attribute_arr[tree_idx];
        for (int i = 0; i < ATTRIBUTE_COUNT_PER_TREE; i++) {
            int val = strtod(arr[cur_attribute_arr[i]].c_str(), NULL) < 0 ? -1 : 1;
            h_data[instance_idx * ATTRIBUTE_COUNT_PER_TREE + i] = val;
        }
        h_data[instance_idx * ATTRIBUTE_COUNT_PER_TREE] = class_code_map[arr[arr.size() - 1]]; // class

        instance_idx++;

        if (instance_idx == INSTANCE_COUNT_PER_TREE) {
            hipMemcpy((void *) d_data, (void *) &h_data, INSTANCE_COUNT_PER_TREE * (ATTRIBUTE_COUNT_PER_TREE + 1) * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy((void *) d_cur_attribute_arr, (void *) &cur_attribute_arr, ATTRIBUTE_COUNT_PER_TREE * sizeof(int), hipMemcpyHostToDevice);

            cout << "launching tree_traversal kernel" << endl;
            
            block_count = FOREST_SIZE;
            thread_count = INSTANCE_COUNT_PER_TREE;
            tree_traversal<<<block_count, thread_count>>>(d_decision_trees,
                    d_cur_attribute_arr,
                    d_data,
                    d_leaf_class,
                    d_leaf_back,
                    ATTRIBUTE_COUNT_PER_TREE);

            cout << "tree_traversal completed" << endl;
            instance_idx = 0;

            cout << "launching counter_increase kernel" << endl;

            block_count = LEAF_COUNT;
            thread_count = ATTRIBUTE_COUNT_PER_TREE * 2;
            // counter_increase<<<block_count, thread_count>>>();

            cout << "counter_increase completed" << endl;

            cout << "lanuching compute_information_gain kernel" << endl;
            
            int *d_info_gain_vals;

            cout << "Allocating info_gain_vals..." << endl;
            err = hipMalloc((void **) &d_info_gain_vals, FOREST_SIZE * LEAF_COUNT * sizeof(float));
            if (err) {
                cout << "error allocating memory for info_gain_vals" <<endl;
                return 1;
            } else {
                cout << "device: memory for info_gain_vals allocated successfully." << endl;
            }

            dim3 grid(FOREST_SIZE, LEAF_COUNT);
            thread_count = ATTRIBUTE_COUNT_PER_TREE * 2;
            compute_information_gain<<<grid, thread_count>>>(d_leaf_counters,
                    d_info_gain_vals,
                    CLASS_COUNT);

            cout << "compute_information_gain completed" << endl;
            
            hipFree(d_info_gain_vals);
        }

        break; // TODO 
    }

    hipFree(d_decision_trees);
    hipFree(d_leaf_class);
    hipFree(d_leaf_back);
    hipFree(d_data);
    hipFree(d_cur_attribute_arr);
    
    return 0;
}
