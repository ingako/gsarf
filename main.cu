#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <limits.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <algorithm>
#include <map>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "ADWIN.cu"
#include "LRU_state.cu"

using namespace std;

#define EPS 1e-5
#define IS_BIT_SET(val, pos) (val & (1 << pos))

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

template <typename T>
bool allocate_memory_on_device(T **arr, string arr_name, int count) {
    size_t memory_size = count * sizeof(T);
    // cout << "\nAllocating " << memory_size << " bytes for " << arr_name << " on device..." << endl;

    hipError_t err = hipMalloc((void **) arr, memory_size); // allocate global memory on the device
    if (err != hipSuccess) {
        // cout << "error allocating memory for " << arr_name << " on device: " << memory_size << " bytes" << endl;
        return false;
    } else {
        hipMemset((void **) arr, 0, memory_size);
        // cout << "device: memory for " << arr_name << " allocated successfully." << endl;
        return true;
    }
}

double get_kappa(int *confusion_matrix, int class_count,  double accuracy, int sample_count) {
    // computes the Cohen's kappa coefficient

    double p0 = accuracy;
    double pc = 0.0;
    int row_count = class_count;
    int col_count = class_count;

    for (int i = 0; i < row_count; i++) {
        double row_sum = 0;
        for (int j = 0; j < col_count; j++) {
            row_sum += confusion_matrix[i * col_count + j];
        }

        double col_sum = 0;
        for (int j = 0; j < row_count; j++) {
            col_sum += confusion_matrix[i * row_count + j];
        }

        pc += (row_sum / sample_count) * (col_sum / sample_count);
    }

    if (pc == 1) {
        return 1;
    }

    return (p0 - pc) / (1.0 - pc);
}



void select_k_attributes(int *reservoir, int n, int k) {
    for (int i = 0; i < k; i++) {
        reservoir[i] = rand() % n;
    }

    // int i;
    // for (i = 0; i < k; i++) {
    //     reservoir[i] = i;
    // }

    // for (i = k; i < n; i++) {
    //     int j = rand() % (i + 1);

    //     if (j < k) reservoir[j] = i;
    // }
}

vector<string> split_attributes(string line, char delim) {
    vector<string> arr;
    const char *start = line.c_str();
    bool instring = false;

    for (const char* p = start; *p; p++) {
        if (*p == '"') {
            instring = !instring;
        } else if (*p == delim && !instring) {
            arr.push_back(string(start, p-start));
            start = p + 1;
        }
    }

    arr.push_back(string(start)); // last field delimited by end of line instead of comma
    return arr;
}

vector<string> split(string str, string delim) {
    char* cstr = const_cast<char*>(str.c_str());
    char* current;
    vector<string> arr;
    current = strtok(cstr, delim.c_str());

    while (current != NULL) {
        arr.push_back(current);
        current = strtok(NULL, delim.c_str());
    }

    return arr;
}

__global__ void setup_kernel(hiprandState *state) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(42, idx, 0, &state[idx]);
}

__device__ unsigned int get_left(unsigned int index) {
    return 2 * index + 1;
}

__device__ unsigned int get_right(unsigned int index) {
    return 2 * index + 2;
}

__device__ int get_rand(int low, int high, hiprandState *local_state) {
    float randu_f = hiprand_uniform(local_state);
    randu_f *= (high - low + 0.999999);
    randu_f += low;
    int randu_int = __float2int_rz(randu_f);

    return randu_int;
}

__device__ int poisson(float lambda, hiprandState *local_state) {
	float product = 1.0;
	float sum = 1.0;

    int rand_num = get_rand(0, 1000, local_state);

	float next_double = (float) rand_num / 1000.0;
	float threshold = next_double * exp(lambda);
	int max_val = max(100, 10 * (int) (lambda));

	int i = 1;
	while ((i < max_val) && (sum <= threshold)) {
		product *= (lambda / i);
		sum += product;
		i++;
	}

	return i - 1;
}

__global__ void reset_tree(
        int *reseted_tree_idx_arr,
        int *decision_trees,
        int *leaf_counters,
        int *leaf_class,
        int *leaf_back,
        int *samples_seen_count,
        int *node_count_per_tree,
        int *leaf_count_per_tree,
        int max_node_count_per_tree,
        int max_leaf_count_per_tree,
        int leaf_counter_size,
        int leaf_counter_row_len,
        int class_count) {

    // <<<1, reseted_tree_count>>>

    if (threadIdx.x >= blockDim.x) {
        return;
    }

    int tree_idx = reseted_tree_idx_arr[threadIdx.x];

    node_count_per_tree[tree_idx] = 1;
    leaf_count_per_tree[tree_idx] = 1;

    int *cur_decision_tree = decision_trees + tree_idx * max_node_count_per_tree;
    int *cur_leaf_class = leaf_class + tree_idx * max_leaf_count_per_tree;
    int *cur_leaf_back = leaf_back + tree_idx * max_leaf_count_per_tree;
    int *cur_samples_seen_count = samples_seen_count + tree_idx * max_leaf_count_per_tree;

    cur_decision_tree[0] = (1 << 31);
    cur_leaf_class[0] = 0;
    cur_leaf_back[0] = 0;

    for (int i = 0; i < max_leaf_count_per_tree; i++) {
        cur_samples_seen_count[i] = 0;
    }

    int *cur_leaf_counter = leaf_counters + tree_idx * max_leaf_count_per_tree * leaf_counter_size;

    for (int k = 0; k < class_count + 2; k++) {
        for (int ij = 0; ij < leaf_counter_row_len; ij++) {
            cur_leaf_counter[k * leaf_counter_row_len + ij] = k == 1 ? 1 : 0;
        }
    }
}

__global__ void tree_traversal(
        int *decision_trees,
        int *tree_status,
        int *data,
        int *reached_leaf_ids,
        int *leaf_class,
        int *correct_counter,
        int *samples_seen_count,
        int *forest_vote,
        int *forest_vote_idx_arr,
        int *weights,
        int *tree_error_count,
        int *confusion_matrix,
        int *class_count_arr,
        int majority_class,
        int node_count_per_tree,
        int leaf_count_per_tree,
        int attribute_count_total,
        int class_count,
        hiprandState *state) {
    // <<<TREE_COUNT, INSTANCE_COUNT_PER_TREE>>>

    int tree_idx = blockIdx.x;

    int cur_tree_status = tree_status[tree_idx];
    if (cur_tree_status == 0 || cur_tree_status == 2) {
        // tree is either inactive or an inactive background tree
        return;
    }

    int instance_idx = threadIdx.x;
    int instance_count_per_tree = blockDim.x;
    int thread_pos = instance_idx + tree_idx * instance_count_per_tree;

    if (thread_pos >= blockDim.x * gridDim.x) {
        return;
    }

    int *cur_data_line = data + instance_idx * (attribute_count_total + 1);
    int *cur_decision_tree = decision_trees + tree_idx * node_count_per_tree;
    int *cur_reached_leaf_ids = reached_leaf_ids + tree_idx * instance_count_per_tree;
    int *cur_leaf_class = leaf_class + tree_idx * leaf_count_per_tree;
    int *cur_samples_seen_count = samples_seen_count + tree_idx * leaf_count_per_tree;
    int *cur_forest_vote = forest_vote + instance_idx * class_count;

    int pos = 0;
    while (!IS_BIT_SET(cur_decision_tree[pos], 31)) {
        int attribute_id = cur_decision_tree[pos];
        pos = cur_data_line[attribute_id] == 0 ? get_left(pos) : get_right(pos);
    }

    int leaf_offset = (cur_decision_tree[pos] & (~(1 << 31)));
    cur_reached_leaf_ids[instance_idx] = leaf_offset;

    atomicAdd(&cur_samples_seen_count[leaf_offset], 1);

    // online bagging
    int *cur_weights = weights + tree_idx * instance_count_per_tree;

    // hiprand library poisson is super slow!
    // cur_weights[instance_idx] = hiprand_poisson(state + thread_pos, 1.0);

    // prepare weights to be used in counter_increase kernel
    cur_weights[instance_idx] = poisson(1.0, state + thread_pos);
    // printf("==================================cur weight: %i\n", cur_weights[instance_idx]);

    if (cur_tree_status == 3) {
        // growing background tree does not particiate in voting
        return;
    }

    int predicted_class = cur_leaf_class[leaf_offset];
    int actual_class = cur_data_line[attribute_count_total];

    if (pos == 0) {
        predicted_class = majority_class;
    }

    if (predicted_class != actual_class) {
        atomicAdd(&tree_error_count[tree_idx], 1);
    }

    if (class_count_arr[predicted_class] == 0) {
        predicted_class = majority_class;
    }

    atomicAdd(&cur_forest_vote[predicted_class], 1);


    __syncthreads();

    if (tree_idx != 0) {
        return;
    }

    int *cur_forest_vote_idx_arr = forest_vote_idx_arr + instance_idx * class_count;

    thrust::sort_by_key(thrust::seq,
            cur_forest_vote,
            cur_forest_vote + class_count,
            cur_forest_vote_idx_arr);

    int voted_class = cur_forest_vote_idx_arr[class_count - 1];

    atomicAdd(&confusion_matrix[actual_class * class_count + voted_class], 1);

    if (voted_class == actual_class) {
        atomicAdd(correct_counter, 1);
    }
}

__global__ void counter_increase(
        int *leaf_counters,
        int *tree_status,
        int *reached_leaf_ids,
        int *data,
        int *weights,
        int class_count,
        int attribute_count_total,
        int leaf_count_per_tree,
        int leaf_counter_size) {
    // gridDim: dim3(TREE_COUNT, INSTANCE_COUNT_PER_TREE)
    // blockDim: ATTRIBUTE_COUNT_TOTAL
    // increment both n_ij (at row 0) and n_ijk (at row k)

    // input: an array of leaf_ids (offset) and leaf_classes built from tree_traversal

    // *** Each leaf counter is represented by a block and uses one thread for each attribute i and
    // value j (i.e. one thread per column)
    //
    // Row 0 stores the total number of times value n_ij appeared.
    // Row 1 is a mask that keeps track of which attributes have been already used in internal nodes
    // along the path.
    // Row 2 and onwards stores partial counters n_ijk for each class k.

    int tree_idx = blockIdx.x;
    int cur_tree_status = tree_status[tree_idx];
    if (cur_tree_status == 0 || cur_tree_status == 2) {
        return;
    }

    int instance_idx = blockIdx.y;
    int instance_count_per_tree = gridDim.y;

    int block_id = blockIdx.y + blockIdx.x * gridDim.y;

    int thread_pos = threadIdx.x + block_id * blockDim.x;
    if (thread_pos >= gridDim.x * gridDim.y * blockDim.x) {
        return;
    }

    int *cur_reached_leaf_ids = reached_leaf_ids + tree_idx * instance_count_per_tree;
    int reached_leaf_id = cur_reached_leaf_ids[instance_idx];

    int *cur_data = data + instance_idx * (attribute_count_total + 1);
    int *cur_weights = weights + tree_idx * instance_count_per_tree;
    int cur_weight = cur_weights[instance_idx];

    // the counter start position corresponds to the leaf_id i.e. leaf offset
    int counter_start_pos = reached_leaf_id * leaf_counter_size + tree_idx *
        leaf_count_per_tree * leaf_counter_size;
    int *cur_leaf_counter = leaf_counters + counter_start_pos;
    // printf("leaf counter start pos is:  %i\n", counter_start_pos);

    int ij = cur_data[threadIdx.x] + threadIdx.x * 2; // binary value 0 or 1
    int k = cur_data[attribute_count_total]; // class

    // int mask = cur_leaf_counter[attribute_count_per_tree * 2 + ij];
    int n_ijk_idx = (k + 2) * attribute_count_total * 2 + ij;

    // atomicAdd(&cur_leaf_counter[ij], mask); // row 0
    // atomicAdd(&cur_leaf_counter[n_ijk_idx], mask);
    atomicAdd(&cur_leaf_counter[ij], cur_weight); // row 0
    atomicAdd(&cur_leaf_counter[n_ijk_idx], cur_weight);
}

__global__ void compute_information_gain(
        int *leaf_counters,
        int *tree_status,
        int *leaf_class,
        float *info_gain_vals,
        int *attribute_val_arr,
        int attribute_count_per_tree,
        int attribute_count_total,
        int class_count,
        int leaf_counter_size) {
    // each leaf_counter is mapped to one block in the 1D grid
    // one thread uses one whole column per leaf counter
    // each block needs as many threads as twice number of the (binary) attributes

    // output: a vector with the attributes information gain values for all leaves in each of the trees
    // gridDim: dim3(TREE_COUNT, LEAF_COUNT_PER_TREE)
    // blockDim: attributes_per_tree * 2 (equal to the length of a info_gain_vals per leaf)

    int block_id = blockIdx.y + blockIdx.x * gridDim.y;

    int thread_pos = threadIdx.x + block_id * blockDim.x;
    if (thread_pos >= gridDim.x * gridDim.y * blockDim.x) {
        return;
    }

    int tree_idx = blockIdx.x;

    int cur_tree_status = tree_status[tree_idx];
    if (cur_tree_status == 0 || cur_tree_status == 2) {
        return;
    }

    int leaf_id = blockIdx.y;

    int leaf_count_per_tree = gridDim.y;
    int leaf_counter_row_len = attribute_count_total * 2;

    int cur_tree_counters_start_pos = tree_idx * leaf_count_per_tree * leaf_counter_size;
    int cur_leaf_counter_start_pos = cur_tree_counters_start_pos + leaf_id * leaf_counter_size;
    int *cur_leaf_counter = leaf_counters + cur_leaf_counter_start_pos;

    int *cur_attribute_val_arr = attribute_val_arr + tree_idx * attribute_count_per_tree;

    int info_gain_per_len = attribute_count_per_tree * 2;
    int cur_tree_info_gain_start_pos = tree_idx * leaf_count_per_tree * info_gain_per_len;
    int cur_leaf_info_gain_start_pos = cur_tree_info_gain_start_pos + leaf_id *
        info_gain_per_len;
    float *cur_info_gain_vals = info_gain_vals + cur_leaf_info_gain_start_pos;

    int col_idx = cur_attribute_val_arr[threadIdx.x / 2] * 2 + threadIdx.x % 2; // TODO expensive mod

    int a_ij = cur_leaf_counter[col_idx];
    int mask = cur_leaf_counter[leaf_counter_row_len + col_idx];
    cur_info_gain_vals[threadIdx.x] = FLT_MAX;


    if (mask == 1) {
        // sum up a column
        float sum = 0.0;

        for (int i = 0; i < class_count; i++) {
            int a_ijk = cur_leaf_counter[col_idx + (2 + i) * leaf_counter_row_len];

            // float param = a_ijk / a_ij; // TODO float division by zero returns INF
            // asm("max.f32 %0, %1, %2;" : "=f"(param) : "f"(param), "f"((float) 0.0));
            // sum += param * log(param);

            float param = 0.0;
            if (a_ijk != 0) { // && a_ij != 0) {
                param = (float) a_ijk / (float) a_ij;
            }

            float log_param = 0.0;
            if (abs(param) > EPS) {
                log_param = log(param);
            }

            sum += param * log_param;
        }

        cur_info_gain_vals[threadIdx.x] = -sum;
    }

    __syncthreads();

    float i_00 = 0.0, i_01 = 0.0;
    int i_idx = 0;

    if (threadIdx.x % 2 == 0) {
        i_00 = cur_info_gain_vals[threadIdx.x];
        i_01 = cur_info_gain_vals[threadIdx.x + 1];
        i_idx = (threadIdx.x >> 1);
    }

    __syncthreads();

    if (threadIdx.x % 2 == 0) {
        cur_info_gain_vals[i_idx] = i_00 + i_01;
    }

    if (threadIdx.x != 0) {
        return;
    }

    int majority_class_code = 0;
    int majority_class_count = 0;

    for (int k = 0; k < class_count; k++) {
        int a_k = cur_leaf_counter[threadIdx.x + (2 + k) * leaf_counter_row_len]
                + cur_leaf_counter[threadIdx.x + 1 + (2 + k) * leaf_counter_row_len];

        if (a_k > majority_class_count) {
            majority_class_count = a_k;
            majority_class_code = k;
        }
    }

    int *cur_leaf_class = leaf_class + tree_idx * leaf_count_per_tree;
    cur_leaf_class[leaf_id] = majority_class_code;
}

// hoeffding bound
// providing an upper bound on the probability that the sum of a sample of independent random
// variables deviates from its expected value
//
// range: range of the random variable
// confidence: desired probability of the estimate not being within the expected value
// n: the number of examples collected at the node
__device__ float compute_hoeffding_bound(float range, float confidence, float n) {
    float result = sqrt(((range * range) * log(1.0 / confidence)) / (2.0 * n));
    // printf("=========> range: %f, confidence: %f, n: %f, result: %f\n", range, confidence, n, result);
    return result;
}

__global__ void compute_node_split_decisions(
        float *info_gain_vals,
        int *tree_status,
        int *attribute_val_arr,
        int *attribute_idx_arr,
        int *node_split_decisions,
        int attribute_count_per_tree,
        float r,
        float delta,
        int *samples_seen_count) {
    // <<<TREE_COUNT, LEAF_COUNT_PER_TREE>>>
    // note: different from paper by using one thread per leaf
    // output: an array of decisions
    //         - the most significant bit denotes whether a leaf needs to be split
    //         - the rest bits denote the attribute id to split on

    int thread_pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_pos >= gridDim.x * blockDim.x) {
        return;
    }

    int tree_idx = blockIdx.x;
    int cur_tree_status = tree_status[tree_idx];
    if (cur_tree_status == 0 || cur_tree_status == 2) {
        return;
    }

    int leaf_idx = threadIdx.x;
    int leaf_count_per_tree = blockDim.x;

    int *cur_attribute_val_arr = attribute_val_arr + tree_idx * attribute_count_per_tree;

    int cur_tree_attr_idx_start_pos = tree_idx * leaf_count_per_tree * attribute_count_per_tree;
    int cur_leaf_attr_idx_start_pos = cur_tree_attr_idx_start_pos + leaf_idx *
        attribute_count_per_tree;
    int *cur_attribute_idx_arr = attribute_idx_arr + cur_leaf_attr_idx_start_pos;

    int cur_tree_info_gain_start_pos = tree_idx * leaf_count_per_tree
        * attribute_count_per_tree * 2;
    int cur_leaf_info_gain_start_pos = cur_tree_info_gain_start_pos + leaf_idx *
        attribute_count_per_tree * 2;
    float *cur_info_gain_vals = info_gain_vals + cur_leaf_info_gain_start_pos;

    thrust::sort_by_key(thrust::seq,
            cur_info_gain_vals,
            cur_info_gain_vals + attribute_count_per_tree,
            cur_attribute_idx_arr);

    float first_best = cur_info_gain_vals[0];
    float second_best = cur_info_gain_vals[1];

    float hoeffding_bound = compute_hoeffding_bound(r, delta, samples_seen_count[thread_pos]);

    int decision = 0;
    if (fabs(first_best - second_best) > hoeffding_bound) {
        // split on the best attribute
        decision |= (1 << 31);
        decision |= cur_attribute_val_arr[cur_attribute_idx_arr[0]];
    }

    node_split_decisions[thread_pos] = decision;
}

__global__ void node_split(
        int *decision_trees,
        int *tree_status,
        int *node_split_decisions,
        int *leaf_counters,
        int *leaf_class,
        int *leaf_back,
        int *attribute_val_arr,
        int *samples_seen_count,
        int *cur_node_count_per_tree,
        int *cur_leaf_count_per_tree,
        int counter_size_per_leaf,
        int max_node_count_per_tree,
        int max_leaf_count_per_tree,
        int attribute_count_per_tree,
        int attribute_count_total,
        int class_count) {
    // <<<1, TREE_COUNT>>>
    // only launch one thread for each tree
    // to append new leaves at the end of the decision_tree array sequentially

    if (threadIdx.x >= blockDim.x) {
        return;
    }

    int tree_idx = threadIdx.x;
    int cur_tree_status = tree_status[tree_idx];

    if (cur_tree_status == 0 || cur_tree_status == 2) {
        // tree is either inactive or an inactive background tree
        return;
    }

    int cur_node_count = cur_node_count_per_tree[tree_idx];
    int cur_leaf_count = cur_leaf_count_per_tree[tree_idx];

    int *cur_decision_tree = decision_trees + tree_idx * max_node_count_per_tree;

    int *cur_node_split_decisions = node_split_decisions + tree_idx *
        max_leaf_count_per_tree;

    int *cur_tree_leaf_counters = leaf_counters +
        tree_idx * max_leaf_count_per_tree * counter_size_per_leaf;

    int *cur_leaf_back = leaf_back + tree_idx * max_leaf_count_per_tree;
    int *cur_leaf_class = leaf_class + tree_idx * max_leaf_count_per_tree;

    int *cur_attribute_val_arr = attribute_val_arr + tree_idx * attribute_count_per_tree;

    for (int leaf_idx = 0; leaf_idx < max_leaf_count_per_tree; leaf_idx++) {
        unsigned int decision = cur_node_split_decisions[leaf_idx];
        cur_node_split_decisions[leaf_idx] = 0;

        int *cur_leaf_counter = cur_tree_leaf_counters + leaf_idx * counter_size_per_leaf;

        if (cur_node_count == max_node_count_per_tree) {
            // tree is full
            return;
        }

        if (!IS_BIT_SET(decision, 31)) {
            continue;
        }

        int attribute_id = (decision & ~(1 << 31));
        int cur_leaf_pos_in_tree = cur_leaf_back[leaf_idx];
        int cur_leaf_val = cur_decision_tree[cur_leaf_pos_in_tree];

        int old_leaf_id = (cur_leaf_val & ~(1 << 31));
        int new_leaf_id = cur_leaf_count;

        int *cur_samples_seen_count = samples_seen_count + tree_idx * max_leaf_count_per_tree;

        cur_samples_seen_count[old_leaf_id] = 0;
        cur_samples_seen_count[new_leaf_id] = 0;

        int left_leaf_pos = get_left(cur_leaf_pos_in_tree);
        int right_leaf_pos = get_right(cur_leaf_pos_in_tree);

        if (left_leaf_pos >= max_node_count_per_tree
                || right_leaf_pos >= max_node_count_per_tree) {
            continue;
        }

        cur_decision_tree[cur_leaf_pos_in_tree] = attribute_id;
        // cur_decision_tree[cur_leaf_pos_in_tree] = cur_attribute_val_arr[attribute_id];

        cur_decision_tree[left_leaf_pos] = cur_leaf_val;
        cur_decision_tree[right_leaf_pos] = (1 << 31) | new_leaf_id;

        cur_leaf_back[old_leaf_id] = left_leaf_pos;
        cur_leaf_back[new_leaf_id] = right_leaf_pos;


        int left_max_class_code = 0;
        int left_max_count = cur_leaf_counter[attribute_count_total * 2 * 2
            + attribute_id * 2];

        int right_max_class_code = 0;
        int right_max_count = cur_leaf_counter[attribute_count_total * 2 * 2
            + attribute_id * 2 + 1];

        for (int k = 1; k < class_count; k++) {
            // left
            int cur_left_class_count = cur_leaf_counter[attribute_count_total * 2 * (k + 2) +
                attribute_id * 2];
            if (cur_left_class_count > left_max_count) {
                left_max_count = cur_left_class_count;
                left_max_class_code = k;
            }

            // right
            int cur_right_class_count = cur_leaf_counter[attribute_count_total * 2 * (k + 2) +
                attribute_id * 2 + 1];
            if (cur_right_class_count > right_max_count) {
                right_max_count = cur_right_class_count;
                right_max_class_code = k;
            }
        }

        cur_leaf_class[old_leaf_id] = left_max_class_code;
        cur_leaf_class[new_leaf_id] = right_max_class_code;


        // reset current leaf_counter and add copy mask to a new leaf counter
        int *new_leaf_counter = cur_tree_leaf_counters + cur_leaf_count * counter_size_per_leaf;

        for (int k = 0; k < class_count + 2; k++) {
            int *cur_leaf_counter_row = cur_leaf_counter + attribute_count_total * 2 * k;
            int *new_leaf_counter_row = new_leaf_counter + attribute_count_total * 2 * k;

            if (k == 1) {
                for (int ij = 0; ij < attribute_count_total * 2; ij++) {
                    if (ij == attribute_id * 2 || ij == attribute_id * 2 + 1) {
                        cur_leaf_counter_row[ij] = 0;
                    }

                    new_leaf_counter_row[ij] = cur_leaf_counter_row[ij];
                }

            } else {
                for (int ij = 0; ij < attribute_count_total * 2; ij++) {
                    cur_leaf_counter_row[ij] = 0;
                    new_leaf_counter_row[ij] = 0;
                }
            }
        }

        cur_node_count += 2;
        cur_leaf_count += 1;
    }

    cur_node_count_per_tree[tree_idx] = cur_node_count;
    cur_leaf_count_per_tree[tree_idx] = cur_leaf_count;
}

int main(int argc, char *argv[]) {

    int TREE_COUNT = 1;
    int TREE_DEPTH_PARAM = -1;
    int INSTANCE_COUNT_PER_TREE = 200;
    int SAMPLE_FREQUENCY = 1000;
    float n_min = 50; // hoeffding bound parameter, grace_period

    string data_path = "data/covtype";
    string data_file_name = "covtype_binary_attributes.csv";

    bool ENABLE_BACKGROUND_TREES = false;

    int opt;
    while ((opt = getopt(argc, argv, "t:i:p:n:s:d:g:br")) != -1) {
        switch (opt) {
            case 't':
                TREE_COUNT = atoi(optarg);
                break;
            case 'i':
                INSTANCE_COUNT_PER_TREE = atoi(optarg);
                break;
            case 'p':
                data_path = optarg;
                break;
            case 'n':
                data_file_name = optarg;
                break;
            case 's':
                SAMPLE_FREQUENCY = atoi(optarg);
                break;
            case 'b':
                ENABLE_BACKGROUND_TREES = true;
                break;
            case 'd':
                TREE_DEPTH_PARAM = atoi(optarg);
                break;
            case 'g':
                n_min = atoi(optarg);
                break;
            case 'r':
                // Use a different seed value for each run
                srand(time(NULL));
                break;
        }
    }

    if (ENABLE_BACKGROUND_TREES) {
        TREE_COUNT *= 2;
    }

    ofstream log_file;
    log_file.open("log_file.txt");

    log_file << "TREE_COUNT = " << TREE_COUNT << endl
        << "INSTANCE_COUNT_PER_TREE = " << INSTANCE_COUNT_PER_TREE << endl;


    string output_path = data_path + "/result_gpu.csv";
    ofstream output_file;
    output_file.open(output_path);

    log_file << endl;
    if (output_file.fail()) {
        log_file << "Error opening output file at " << output_path << endl;
        return 1;
    } else {
        log_file << "Writing output to " << output_path << endl;
    }


    // read data file
    string attribute_file_path = data_path + "/attributes.txt";
    ifstream attribute_file(attribute_file_path);

    log_file << endl;
    if (attribute_file) {
        log_file << "Reading data file from " << attribute_file_path << " succeeded." << endl;
    } else {
        log_file << "Error reading file from " << attribute_file_path << endl;
        return 1;
    }

    // prepare attributes
    string line;
    getline(attribute_file, line);

    const int ATTRIBUTE_COUNT_TOTAL = split(line, ",").size() - 1;
    const int ATTRIBUTE_COUNT_PER_TREE = (int) sqrt(ATTRIBUTE_COUNT_TOTAL);

    const int TREE_DEPTH =
        TREE_DEPTH_PARAM == -1 ? (int) sqrt(ATTRIBUTE_COUNT_TOTAL) + 1 : TREE_DEPTH_PARAM;

    log_file << "ATTRIBUTE_COUNT_TOTAL = " << ATTRIBUTE_COUNT_TOTAL << endl;
    log_file << "ATTRIBUTE_COUNT_PER_TREE = " << ATTRIBUTE_COUNT_PER_TREE << endl;
    log_file << "TREE_DEPTH = " << TREE_DEPTH << endl;

    const unsigned int NODE_COUNT_PER_TREE = (1 << TREE_DEPTH) - 1;
    const unsigned int LEAF_COUNT_PER_TREE = (1 << (TREE_DEPTH - 1));

    log_file << "NODE_COUNT_PER_TREE = " << NODE_COUNT_PER_TREE << endl;
    log_file << "LEAF_COUNT_PER_TREE = " << LEAF_COUNT_PER_TREE << endl;


    // read class/label file
    string class_path = data_path + "/labels.txt";
    ifstream class_file(class_path);

    log_file << endl;
    if (class_file) {
        log_file << "Reading class file from " << class_path << " succeeded." << endl;
    } else {
        log_file << "Error reading class file from " << class_path << endl;
    }

    string class_line;

    // init mapping between class and code
    map<string, int> class_code_map;
    map<int, string> code_class_map;

    vector<string> class_arr = split(class_line, " ");
    string code_str, class_str;

    int line_count = 0;
    while (class_file >> class_str) {
        int class_code = line_count;
        class_code_map[class_str] = class_code;
        code_class_map[class_code] = class_str;
        line_count++;
    }
    const int CLASS_COUNT = line_count;
    log_file << "CLASS_COUNT = " << CLASS_COUNT << endl;

    // hoeffding bound parameters
    float delta = 0.05; // pow((float) 10.0, -7);
    float r = log2(CLASS_COUNT); // range of merit = log2(num_of_classes)

    log_file << endl
        << "hoeffding bound parameters: " << endl
        << "n_min = " << n_min << endl
        << "delta = " << delta << endl
        << "r     = " << r     << endl;


    // init decision tree
    log_file << "\nAllocating memory on host..." << endl;
    // void *allocated = malloc(NODE_COUNT_PER_TREE * TREE_COUNT * sizeof(int));
    void *allocated = calloc(NODE_COUNT_PER_TREE * TREE_COUNT, sizeof(int)); // TODO
    if (allocated == NULL) {
        log_file << "host error: memory allocation for decision trees failed" << endl;
        return 1;
    }
    int *h_decision_trees = (int*) allocated;

    int *d_decision_trees;
    if (!allocate_memory_on_device(&d_decision_trees, "decision_trees", NODE_COUNT_PER_TREE * TREE_COUNT)) {
        return 1;
    }


#if DEBUG

    allocated = malloc(LEAF_COUNT_PER_TREE * TREE_COUNT * sizeof(int));
    if (allocated == NULL) {
        log_file << "host error: memory allocation for leaf_class failed" << endl;
        return 1;
    }
    int *h_leaf_class = (int*) allocated; // stores the class for a given leaf

    allocated = malloc(LEAF_COUNT_PER_TREE * TREE_COUNT * sizeof(int));
    if (allocated == NULL) {
        log_file << "host error: memory allocation for leaf_back failed" << endl;
        return 1;
    }
    int *h_leaf_back = (int*) allocated; // reverse pointer to map a leaf id to an offset in the tree array

#endif

    log_file << "Init: set root as leaf for each tree in the forest..." << endl;
    for (int i = 0; i < TREE_COUNT; i++) {
        int *cur_decision_tree = h_decision_trees + i * NODE_COUNT_PER_TREE;
        cur_decision_tree[0] = (1 << 31);

        for (int j = 1; j < NODE_COUNT_PER_TREE; j++) {
            cur_decision_tree[j] = -1;
        }

    }

    gpuErrchk(hipMemcpy(d_decision_trees, h_decision_trees, NODE_COUNT_PER_TREE * TREE_COUNT
                * sizeof(int), hipMemcpyHostToDevice));

    // the offsets of leaves reached from tree traversal
    int *d_reached_leaf_ids;
    if (!allocate_memory_on_device(&d_reached_leaf_ids, "leaf_ids", INSTANCE_COUNT_PER_TREE * TREE_COUNT)) {
        return 1;
    }

    int *d_leaf_class;
    if (!allocate_memory_on_device(&d_leaf_class, "leaf_class", LEAF_COUNT_PER_TREE * TREE_COUNT)) {
        return 1;
    }

    int *d_leaf_back;
    if (!allocate_memory_on_device(&d_leaf_back, "leaf_back", LEAF_COUNT_PER_TREE * TREE_COUNT)) {
        return 1;
    }

    // TODO: for testing only
    int leaf_counter_size = ATTRIBUTE_COUNT_TOTAL * 2 * (CLASS_COUNT + 2);
    int all_leaf_counters_size = TREE_COUNT * LEAF_COUNT_PER_TREE * leaf_counter_size;

    // int *h_leaf_counters = (int*) malloc(all_leaf_counters_size * sizeof(int));
    int *h_leaf_counters = (int*) calloc(all_leaf_counters_size, sizeof(int));

    // init mask row
    for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
        int *cur_tree_leaf_counters = h_leaf_counters + tree_idx * LEAF_COUNT_PER_TREE *
            leaf_counter_size;
        for (int leaf_idx = 0; leaf_idx < LEAF_COUNT_PER_TREE; leaf_idx++) {
            int *cur_leaf_counter = cur_tree_leaf_counters + leaf_idx * leaf_counter_size;
            int *cur_leaf_counter_mask_row = cur_leaf_counter + ATTRIBUTE_COUNT_TOTAL * 2;

            for (int k = 0; k < ATTRIBUTE_COUNT_TOTAL * 2; k++) {
                cur_leaf_counter_mask_row[k] = 1;
            }
        }
    }

    int *d_leaf_counters;
    if (!allocate_memory_on_device(&d_leaf_counters, "leaf_counters", all_leaf_counters_size)) {
        return 1;
    }
    gpuErrchk(hipMemcpy(d_leaf_counters, h_leaf_counters, all_leaf_counters_size * sizeof(int),
                hipMemcpyHostToDevice));

    // TODO: h_info_gain_vals for testing only
    int info_gain_vals_len = TREE_COUNT * LEAF_COUNT_PER_TREE * ATTRIBUTE_COUNT_PER_TREE * 2;
    float *h_info_gain_vals = (float*) malloc(info_gain_vals_len * sizeof(float));

    float *d_info_gain_vals;
    if (!allocate_memory_on_device(&d_info_gain_vals, "info_gain_vals", info_gain_vals_len)) {
        return 1;
    }


    // actual selected attributes for each tree for counter_increase kernel
    int *h_attribute_val_arr;
    int *d_attribute_val_arr;
    int attribute_val_arr_len = TREE_COUNT * ATTRIBUTE_COUNT_PER_TREE;

    allocated = malloc(attribute_val_arr_len * sizeof(int));
    if (allocated == NULL) {
        log_file << "host error: memory allocation for h_attribute_val_arr failed" << endl;
    }
    h_attribute_val_arr = (int*) allocated;

    if (!allocate_memory_on_device(&d_attribute_val_arr, "attribute_val_arr",
                attribute_val_arr_len)) {
        return 1;
    }

    // allocate memory for attribute indices on host for computing information gain
    int *h_attribute_idx_arr;
    int *d_attribute_idx_arr;
    int attribute_idx_arr_len = TREE_COUNT * LEAF_COUNT_PER_TREE * ATTRIBUTE_COUNT_PER_TREE;

    allocated = malloc(attribute_idx_arr_len * sizeof(int));
    if (allocated == NULL) {
        log_file << "host error: memory allocation for h_attribute_idx_arr failed" << endl;
        return 1;
    }
    h_attribute_idx_arr = (int*) allocated;

    if (!allocate_memory_on_device(&d_attribute_idx_arr, "attribute_idx_arr",
                attribute_idx_arr_len)) {
        return 1;
    }

    for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
        int *cur_tree_attribute_idx_arr = h_attribute_idx_arr + tree_idx * LEAF_COUNT_PER_TREE
            * ATTRIBUTE_COUNT_PER_TREE;

        for (int leaf_idx = 0; leaf_idx < LEAF_COUNT_PER_TREE; leaf_idx++) {
            int *cur_attribute_idx_arr = cur_tree_attribute_idx_arr + leaf_idx *
                ATTRIBUTE_COUNT_PER_TREE;

            for (int i = 0; i < ATTRIBUTE_COUNT_PER_TREE; i++) {
                cur_attribute_idx_arr[i] = i;
            }
        }
    }

    // TODO same attribute_idx_arr for foreground and its background tree

    // allocate memory for node_split_decisions
    // unsigned int *h_node_split_decisions;
    int *d_node_split_decisions;
    int node_split_decisions_len = LEAF_COUNT_PER_TREE * TREE_COUNT;

    // allocated = malloc(node_split_decisions_len * sizeof(unsigned int));
    // if (allocated == NULL) {
    //     log_file << "host error: memory allocation for h_node_split_decisions failed" << endl;
    //     return 1;
    // }
    // h_node_split_decisions = (unsigned int*) allocated;

    if (!allocate_memory_on_device(&d_node_split_decisions, "node_split_decisions",
                node_split_decisions_len)) {
        return 1;
    }

    int samples_seen_count_len = TREE_COUNT * LEAF_COUNT_PER_TREE;
    int *h_samples_seen_count = (int*) calloc(samples_seen_count_len, sizeof(int));
    int *d_samples_seen_count;
    if (!allocate_memory_on_device(&d_samples_seen_count, "samples_seen_count",
                samples_seen_count_len)) {
        return 1;
    }

    int h_cur_node_count_per_tree[TREE_COUNT];
    int *d_cur_node_count_per_tree;

    fill_n(h_cur_node_count_per_tree, TREE_COUNT, 1);

    if (!allocate_memory_on_device(&d_cur_node_count_per_tree, "cur_node_count_per_tree",
                TREE_COUNT)) {
        return 1;
    }
    gpuErrchk(hipMemcpy(d_cur_node_count_per_tree, h_cur_node_count_per_tree,
                TREE_COUNT * sizeof(int), hipMemcpyHostToDevice));

    int h_cur_leaf_count_per_tree[TREE_COUNT];
    int *d_cur_leaf_count_per_tree;

    fill_n(h_cur_leaf_count_per_tree, TREE_COUNT, 1);

    if (!allocate_memory_on_device(&d_cur_leaf_count_per_tree, "leaf_count_per_tree", TREE_COUNT)) {
        return 1;
    }
    gpuErrchk(hipMemcpy(d_cur_leaf_count_per_tree, h_cur_leaf_count_per_tree,
                 TREE_COUNT * sizeof(int), hipMemcpyHostToDevice));

    int forest_vote_len = INSTANCE_COUNT_PER_TREE * CLASS_COUNT;
    int *d_forest_vote;
    if (!allocate_memory_on_device(&d_forest_vote, "forest_vote", forest_vote_len)) {
        return 1;
    }

    int h_forest_vote_idx_arr[forest_vote_len];
    for (int i = 0; i < INSTANCE_COUNT_PER_TREE; i++) {
        for (int j = 0; j < CLASS_COUNT; j++) {
            h_forest_vote_idx_arr[i * CLASS_COUNT + j] = j;
        }
    }
    int *d_forest_vote_idx_arr;
    if (!allocate_memory_on_device(&d_forest_vote_idx_arr, "forest_vote_idx_arr",
                forest_vote_len)) {
        return 1;
    }

    int *d_weights;
    if (!allocate_memory_on_device(&d_weights, "weights", TREE_COUNT * INSTANCE_COUNT_PER_TREE)) {
        return 1;
    }

    // one warning and drift detector per tree to monitor accuracy
    // initialized with the default construct where delta=0.001
    vector<ADWIN> warning_detectors(TREE_COUNT);
    vector<ADWIN> drift_detectors(TREE_COUNT);

    if (ENABLE_BACKGROUND_TREES) {
        for (int i = 0; i < TREE_COUNT; i++) {
            warning_detectors[i] = ADWIN((double) 0.001);
            drift_detectors[i] = ADWIN((double) 0.00001);
        }
    }

    int tree_error_count_len = ENABLE_BACKGROUND_TREES ? TREE_COUNT >> 1 : TREE_COUNT;
    int* h_tree_error_count = (int*) calloc(tree_error_count_len, sizeof(int));
    int* d_tree_error_count;
    if (!allocate_memory_on_device(&d_tree_error_count, "tree_error_count", tree_error_count_len)) {
        return 1;
    }

    int* d_drift_tree_idx_arr;
    if (!allocate_memory_on_device(&d_drift_tree_idx_arr, "reseted_tree_idx_arr", TREE_COUNT)) {
        return 1;
    }

    // pointer to the start of the background decision trees
    int *d_backgound_decision_trees = d_decision_trees + (TREE_COUNT >> 1) * NODE_COUNT_PER_TREE;


    // for swapping background trees when drift is detected
    LRU_state* state_queue = new LRU_state(3, 0);

    // TODO
    // 0: inactive, 1: active, 2: must be inactive
    // add initial state
    vector<char> cur_state(TREE_COUNT);

    for (int i = 0; i < (TREE_COUNT >> 1); i++) {
        cur_state[i] = '1';
    }

    for (int i = (TREE_COUNT >> 1); i < TREE_COUNT; i++) {
        cur_state[i] = ENABLE_BACKGROUND_TREES ? '0' : '1';
    }

    cout << "initial state: ";
    for (int i = 0; i < cur_state.size(); i++) {
        cout << cur_state[i];
    }
    cout << endl;

    state_queue->get(cur_state); // TODO improve LRU_state API


    // TODO
    // 0: inactive, 1: active, 2: ungrown bg_tree, 3: growing bg_tree
    int h_tree_active_status[TREE_COUNT];
    int *d_tree_active_status;
    if (!allocate_memory_on_device(&d_tree_active_status, "d_tree_active_status", TREE_COUNT)) {
        return 1;
    }

    if (ENABLE_BACKGROUND_TREES) {
        for (int i = 0; i < (TREE_COUNT >> 1); i++) {
            h_tree_active_status[i] = 1;
        }
        for (int i = (TREE_COUNT >> 1); i < TREE_COUNT; i++) {
            h_tree_active_status[i] = 2;
        }

    } else {
        for (int i = 0; i < TREE_COUNT; i++) {
            h_tree_active_status[i] = 1;
        }
    }

    cout << "tree active status: ";
    for (int i = 0; i < TREE_COUNT; i++) {
        cout << h_tree_active_status[i] << " ";
    }
    cout << endl;;

    gpuErrchk(hipMemcpy(d_tree_active_status, h_tree_active_status,
                TREE_COUNT * sizeof(int), hipMemcpyHostToDevice));


    // for calculating kappa measurements
    int confusion_matrix_size = CLASS_COUNT * CLASS_COUNT;
    int *h_confusion_matrix = (int*) malloc(confusion_matrix_size * sizeof(int));
    int *d_confusion_matrix;
    if (!allocate_memory_on_device(&d_confusion_matrix, "d_confusion_matrix",
                confusion_matrix_size)) {
        return 1;
    }


    log_file << "\nInitializing training data arrays..." << endl;

    int data_len = INSTANCE_COUNT_PER_TREE * (ATTRIBUTE_COUNT_TOTAL + 1);
    int *h_data = (int*) malloc(data_len * sizeof(int));

    int *d_data;
    if (!allocate_memory_on_device(&d_data, "data", data_len)) {
        return 1;
    }

    int *d_class_count_arr;
    if (!allocate_memory_on_device(&d_class_count_arr, "class_count_arr", CLASS_COUNT)) {
        return 1;
    }

    // read data file
    string csv_path = data_path + "/" + data_file_name;
    ifstream data_file(csv_path);

    log_file << endl;
    if (data_file) {
        log_file << "Reading data file from " << csv_path << " succeeded." << endl;
    } else {
        log_file << "Error reading file from " << csv_path << endl;
        return 1;
    }

    vector<string> raw_data_row;

    int block_count;
    int thread_count;

    log_file << endl << "=====Training Start=====" << endl;

    int h_correct_counter = 0;
    int *d_correct_counter;
    gpuErrchk(hipMalloc((void **) &d_correct_counter, sizeof(int)));

    hiprandState *d_state;
    hipMalloc(&d_state, TREE_COUNT * INSTANCE_COUNT_PER_TREE * sizeof(hiprandState));

    setup_kernel<<<TREE_COUNT, INSTANCE_COUNT_PER_TREE>>>(d_state);
    hipDeviceSynchronize();

    int leaf_counter_row_len = ATTRIBUTE_COUNT_TOTAL * 2;
    int iter_count = 1;

    int sample_count_iter = 0;
    int sample_count_total = 0;
    double window_accuracy = 0.0;
    double window_kappa = 0.0;

    // output_file << "#iteration,accuracy,mean_accuracy,kappa,mean_kappa" << endl;
    output_file << "#iteration,accuracy,kappa" << endl;

    bool eof = false;

    while (!eof) {

        int h_data_idx = 0;
        int class_count_arr[CLASS_COUNT] = { 0 };

        for (int instance_idx = 0; instance_idx < INSTANCE_COUNT_PER_TREE; instance_idx++) {
            if (!getline(data_file, line)) {
                eof = true;
                break;
            }

            raw_data_row = split(line, ",");

            for (int i = 0; i < ATTRIBUTE_COUNT_TOTAL; i++) {
                int val = stoi(raw_data_row[i]);
                h_data[h_data_idx++] = val;
            }

            int cur_class_code = class_code_map[raw_data_row[ATTRIBUTE_COUNT_TOTAL]];
            h_data[h_data_idx] = cur_class_code;
            class_count_arr[cur_class_code]++;

            h_data_idx++;
        }

        if (eof) {
            log_file << "\ntraining completed" << endl;
            break; // TODO
        }

        log_file << endl << "=================iteration " << iter_count
            << "=================" << endl;

        int majority_class = 0;
        int majority_class_count = 0;

        for (int i = 0; i < CLASS_COUNT; i++) {
            if (majority_class_count < class_count_arr[i]) {
                majority_class_count = class_count_arr[i];
                majority_class = i;
            }
        }

        gpuErrchk(hipMemcpy((void *) d_data, (void *) h_data, data_len
                    * sizeof(int), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy((void *) d_class_count_arr, (void *) class_count_arr, CLASS_COUNT
                    * sizeof(int), hipMemcpyHostToDevice));

        log_file << "\nlaunching tree_traversal kernel..." << endl;

        block_count = TREE_COUNT;
        thread_count = INSTANCE_COUNT_PER_TREE;

        gpuErrchk(hipMemset(d_correct_counter, 0, sizeof(int)));
        gpuErrchk(hipMemset(d_tree_error_count, 0, tree_error_count_len * sizeof(int)));
        gpuErrchk(hipMemset(d_confusion_matrix, 0, confusion_matrix_size * sizeof(int)));

        gpuErrchk(hipMemset(d_forest_vote, 0, forest_vote_len * sizeof(int)));
        gpuErrchk(hipMemcpy(d_forest_vote_idx_arr, h_forest_vote_idx_arr, forest_vote_len *
                    sizeof(int), hipMemcpyHostToDevice));

        log_file << "launching " << block_count * thread_count << " threads for tree_traversal" << endl;

        tree_traversal<<<block_count, thread_count>>>(
                d_decision_trees,
                d_tree_active_status,
                d_data,
                d_reached_leaf_ids,
                d_leaf_class,
                d_correct_counter,
                d_samples_seen_count,
                d_forest_vote,
                d_forest_vote_idx_arr,
                d_weights,
                d_tree_error_count,
                d_confusion_matrix,
                d_class_count_arr,
                majority_class,
                NODE_COUNT_PER_TREE,
                LEAF_COUNT_PER_TREE,
                ATTRIBUTE_COUNT_TOTAL,
                CLASS_COUNT,
                d_state);

#if DEBUG

        gpuErrchk(hipMemcpy(h_decision_trees, d_decision_trees, TREE_COUNT * NODE_COUNT_PER_TREE *
                    sizeof(int), hipMemcpyDeviceToHost));

        gpuErrchk(hipMemcpy(h_leaf_class, d_leaf_class, TREE_COUNT * LEAF_COUNT_PER_TREE *
                    sizeof(int), hipMemcpyDeviceToHost));

        gpuErrchk(hipMemcpy((void *) h_samples_seen_count, (void *) d_samples_seen_count,
                    samples_seen_count_len * sizeof(int), hipMemcpyDeviceToHost));

#endif

        hipDeviceSynchronize();
        log_file << "tree_traversal completed" << endl;

        gpuErrchk(hipMemcpy(&h_correct_counter, d_correct_counter, sizeof(int),
                    hipMemcpyDeviceToHost));

        log_file << "h_correct_counter: " << h_correct_counter << endl;

        double accuracy = (double) h_correct_counter / INSTANCE_COUNT_PER_TREE;
        window_accuracy = (sample_count_iter * window_accuracy + accuracy)
            / (sample_count_iter + 1);

        gpuErrchk(hipMemcpy(h_confusion_matrix, d_confusion_matrix,
                    confusion_matrix_size * sizeof(int), hipMemcpyDeviceToHost));

        double kappa = get_kappa(h_confusion_matrix, CLASS_COUNT, accuracy,
                INSTANCE_COUNT_PER_TREE);
        window_kappa = (sample_count_iter * window_kappa + kappa) / (sample_count_iter + 1);

        log_file << "\n=================statistics" << endl
            << "accuracy: " << accuracy << endl
            << "kappa: " << kappa << endl;

        sample_count_iter++;;
        sample_count_total = sample_count_iter * INSTANCE_COUNT_PER_TREE; // avoid expensive mod

        if (sample_count_total >= SAMPLE_FREQUENCY) {
            output_file << iter_count * INSTANCE_COUNT_PER_TREE
                << "," << window_accuracy * 100
                << "," << window_kappa * 100 << endl;

            sample_count_iter = 0;
            window_accuracy = 0.0;
            window_kappa = 0.0;
        }


#if DEBUG

        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
            log_file << "tree " << tree_idx << endl;
            int *cur_decision_tree = h_decision_trees + tree_idx * NODE_COUNT_PER_TREE;
            int *cur_leaf_class = h_leaf_class + tree_idx * LEAF_COUNT_PER_TREE;
            int *cur_samples_seen_count = h_samples_seen_count + tree_idx * LEAF_COUNT_PER_TREE;

            for (int i = 0; i < NODE_COUNT_PER_TREE; i++) {
                log_file << cur_decision_tree[i] << " ";
            }
            log_file << endl;

            for (int i = 0; i < LEAF_COUNT_PER_TREE; i++) {
                log_file << cur_leaf_class[i] << " ";
            }
            log_file << endl;

            log_file << "samples seen count: " << endl;
            for (int i = 0; i < LEAF_COUNT_PER_TREE; i++) {
                log_file << cur_samples_seen_count[i] << " ";
            }
            log_file << endl;
        }

#endif


        log_file << "\nlaunching counter_increase kernel..." << endl;

        counter_increase
            <<<dim3(TREE_COUNT, INSTANCE_COUNT_PER_TREE), ATTRIBUTE_COUNT_TOTAL>>>(
                    d_leaf_counters,
                    d_tree_active_status,
                    d_reached_leaf_ids,
                    d_data,
                    d_weights,
                    CLASS_COUNT,
                    ATTRIBUTE_COUNT_TOTAL,
                    LEAF_COUNT_PER_TREE,
                    leaf_counter_size);

        hipDeviceSynchronize();
        log_file << "counter_increase completed" << endl;

#if DEBUG

        gpuErrchk(hipMemcpy(h_leaf_counters, d_leaf_counters, all_leaf_counters_size
                    * sizeof(int), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(h_cur_leaf_count_per_tree, d_cur_leaf_count_per_tree, TREE_COUNT
                    * sizeof(int), hipMemcpyDeviceToHost));


        log_file << "counter_increase result: " << endl;
        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
            log_file << "tree " << tree_idx << endl;

            log_file << "h_cur_leaf_count_per_tree is: " << h_cur_leaf_count_per_tree[tree_idx] << endl;
            int *cur_tree_leaf_counter = h_leaf_counters + tree_idx * LEAF_COUNT_PER_TREE
                * leaf_counter_size;

            for (int leaf_idx = 0; leaf_idx < h_cur_leaf_count_per_tree[tree_idx]; leaf_idx++) {
                int *cur_leaf_counter = cur_tree_leaf_counter + leaf_idx * leaf_counter_size;
                for (int k = 0; k < CLASS_COUNT + 2; k++) {
                    log_file << "row " << k << ": ";
                    for (int ij = 0; ij < leaf_counter_row_len; ij++) {
                        log_file << right << setw(8)
                            << cur_leaf_counter[k * leaf_counter_row_len + ij] << " ";
                    }
                    log_file << endl;
                }
            }
            log_file << endl;
        }

#endif

        log_file << "\nlanuching compute_information_gain kernel..." << endl;


        // select k random attributes for each tree
        // output_file << "\nAttributes selected per tree: " << endl;

        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {

            // select random attributes for foreground trees only
            if (h_tree_active_status[tree_idx] != 1) {
                continue;
            }

            // output_file << "tree " << tree_idx << endl;

            int *cur_attribute_val_arr = h_attribute_val_arr + tree_idx * ATTRIBUTE_COUNT_PER_TREE;
            select_k_attributes(cur_attribute_val_arr, ATTRIBUTE_COUNT_TOTAL, ATTRIBUTE_COUNT_PER_TREE);

            // for (int i = 0; i < ATTRIBUTE_COUNT_PER_TREE; i++) {
            //     output_file << cur_attribute_val_arr[i] << " ";
            // }
            // output_file << endl;
        }

        gpuErrchk(hipMemcpy(d_attribute_val_arr, h_attribute_val_arr,
                    attribute_val_arr_len * sizeof(int), hipMemcpyHostToDevice));

        // for sorting information gain array
        gpuErrchk(hipMemcpy(d_attribute_idx_arr, h_attribute_idx_arr, attribute_idx_arr_len *
                    sizeof(int), hipMemcpyHostToDevice));


        dim3 grid(TREE_COUNT, LEAF_COUNT_PER_TREE);
        thread_count = ATTRIBUTE_COUNT_PER_TREE * 2;

        compute_information_gain<<<grid, thread_count>>>(
                d_leaf_counters,
                d_tree_active_status,
                d_leaf_class,
                d_info_gain_vals,
                d_attribute_val_arr,
                ATTRIBUTE_COUNT_PER_TREE,
                ATTRIBUTE_COUNT_TOTAL,
                CLASS_COUNT,
                leaf_counter_size);

        hipDeviceSynchronize();
        log_file << "compute_information_gain completed" << endl;



        gpuErrchk(hipMemcpy(d_attribute_idx_arr, h_attribute_idx_arr, attribute_idx_arr_len *
                    sizeof(int), hipMemcpyHostToDevice));

        log_file << "\nlaunching compute_node_split_decisions kernel..." << endl;

        compute_node_split_decisions<<<TREE_COUNT, LEAF_COUNT_PER_TREE>>>(
                d_info_gain_vals,
                d_tree_active_status,
                d_attribute_val_arr,
                d_attribute_idx_arr,
                d_node_split_decisions,
                ATTRIBUTE_COUNT_PER_TREE,
                r,
                delta,
                d_samples_seen_count);

#if DEBUG

        // log info_gain_vals
        gpuErrchk(hipMemcpy(h_info_gain_vals, d_info_gain_vals, info_gain_vals_len *
                    sizeof(float), hipMemcpyDeviceToHost));

        for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
            log_file << "tree " << tree_idx << endl;
            int cur_tree_info_gain_vals_start_pos = tree_idx * LEAF_COUNT_PER_TREE *
                ATTRIBUTE_COUNT_PER_TREE * 2;

            for (int leaf_idx = 0; leaf_idx < LEAF_COUNT_PER_TREE; leaf_idx++) {
                int cur_info_gain_vals_start_pos = cur_tree_info_gain_vals_start_pos + leaf_idx *
                    ATTRIBUTE_COUNT_PER_TREE * 2;
                float *cur_info_gain_vals = h_info_gain_vals + cur_info_gain_vals_start_pos;

                for (int i = 0; i < ATTRIBUTE_COUNT_PER_TREE; i++) {
                    log_file << cur_info_gain_vals[i] << " ";
                }
                log_file << endl;
            }
            log_file << endl;
        }

#endif

        hipDeviceSynchronize();
        log_file << "compute_node_split_decisions completed" << endl;


        log_file << "\nlaunching node_split kernel..." << endl;

        node_split<<<1, TREE_COUNT>>>(
                d_decision_trees,
                d_tree_active_status,
                d_node_split_decisions,
                d_leaf_counters,
                d_leaf_class,
                d_leaf_back,
                d_attribute_val_arr,
                d_samples_seen_count,
                d_cur_node_count_per_tree,
                d_cur_leaf_count_per_tree,
                leaf_counter_size,
                NODE_COUNT_PER_TREE,
                LEAF_COUNT_PER_TREE,
                ATTRIBUTE_COUNT_PER_TREE,
                ATTRIBUTE_COUNT_TOTAL,
                CLASS_COUNT);

        hipDeviceSynchronize();

        log_file << "node_split completed" << endl;

        // for drift detection
        gpuErrchk(hipMemcpy((void *) h_tree_error_count, (void *) d_tree_error_count,
                    tree_error_count_len * sizeof(int), hipMemcpyDeviceToHost));

        int warning_tree_count = 0;
        int drift_tree_count = 0;
        int h_drift_tree_idx_arr[TREE_COUNT];

        // warning/drift detection only on foreground trees
        // if accuracy decreases, reset the tree

        int detection_range = TREE_COUNT;
        if (ENABLE_BACKGROUND_TREES) {
            detection_range = (TREE_COUNT >> 1);
        }

        for (int tree_idx = 0; tree_idx < detection_range; tree_idx++) {

            ADWIN *warning_detector = &warning_detectors[tree_idx];
            double old_error = warning_detector->getEstimation();
            bool error_change = warning_detector->setInput(h_tree_error_count[tree_idx]);

            if (error_change && old_error > warning_detector->getEstimation()) {
                error_change = false;
            }

            if (error_change) {
                warning_tree_count++;
                // warning_detector->resetChange();

                // grow background tree
                int bg_tree_pos = tree_idx + (TREE_COUNT >> 1);
                if (h_tree_active_status[bg_tree_pos] == 2) {
                    // start growing if never grown
                    h_tree_active_status[bg_tree_pos] = 3;
                }

            }

            ADWIN *drift_detector = &drift_detectors[tree_idx];
            old_error = drift_detector->getEstimation();
            error_change = drift_detector->setInput(h_tree_error_count[tree_idx]);

            if (error_change && old_error > drift_detector->getEstimation()) {
                // if error is decreasing, do nothing
                error_change = false;
            }

            if (!error_change) {
                continue;
            }

            warning_detector->resetChange();
            drift_detector->resetChange();

            h_drift_tree_idx_arr[drift_tree_count] = tree_idx;
            drift_tree_count++;
        }

        if (warning_tree_count > 0) {
                cout << endl
                    << "ಠ_ಠ Warning detected at iter_count = " << iter_count << endl
                    << "#warning = " << warning_tree_count << endl;
        }

        if (drift_tree_count > 0) {
            cout << endl
                << "(╯°□°）╯︵ ┻━┻ drift detected at iter_count = " << iter_count << endl
                << "#drift = " << drift_tree_count << endl;

            gpuErrchk(hipMemcpy(d_drift_tree_idx_arr, h_drift_tree_idx_arr,
                        drift_tree_count * sizeof(int), hipMemcpyHostToDevice));

            reset_tree<<<1, drift_tree_count>>>(
                    d_drift_tree_idx_arr,
                    d_decision_trees,
                    d_leaf_counters,
                    d_leaf_class,
                    d_leaf_back,
                    d_samples_seen_count,
                    d_cur_node_count_per_tree,
                    d_cur_leaf_count_per_tree,
                    NODE_COUNT_PER_TREE,
                    LEAF_COUNT_PER_TREE,
                    leaf_counter_size,
                    leaf_counter_row_len,
                    CLASS_COUNT);

            hipDeviceSynchronize();
        }

        iter_count++;
    }

#if DEBUG

    int *h_decision_trees_log = (int*) malloc(NODE_COUNT_PER_TREE * TREE_COUNT * sizeof(int));
    gpuErrchk(hipMemcpy(h_decision_trees_log, d_decision_trees, TREE_COUNT
                * NODE_COUNT_PER_TREE * sizeof(int), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(h_leaf_class, d_leaf_class, TREE_COUNT * LEAF_COUNT_PER_TREE *
                sizeof(int), hipMemcpyDeviceToHost));

    int root_count = 0;
    for (int tree_idx = 0; tree_idx < TREE_COUNT; tree_idx++) {
       cout << "Tree #" << tree_idx << endl;
       int *cur_decision_trees_log = h_decision_trees_log + tree_idx
           * NODE_COUNT_PER_TREE;
       int *cur_leaf_class = h_leaf_class + tree_idx * LEAF_COUNT_PER_TREE;


       for (int i = 0; i < NODE_COUNT_PER_TREE; i++) {
           int val = cur_decision_trees_log[i];
           if (IS_BIT_SET(val, 31) && val != -1) {
               if (i == 0) root_count++;
               int index = (val & (~(1 << 31)));
               val = cur_leaf_class[index];
               cout << "leaf:" << val << " ";
           } else {
               cout << val + 1 << " ";
           }
       }
       cout << endl;
    }

#endif

    hipFree(d_decision_trees);
    hipFree(d_reached_leaf_ids);
    hipFree(d_leaf_class);
    hipFree(d_leaf_back);
    hipFree(d_leaf_counters);
    hipFree(d_data);
    hipFree(d_info_gain_vals);
    hipFree(d_node_split_decisions);
    hipFree(d_samples_seen_count);
    hipFree(d_cur_node_count_per_tree);
    hipFree(d_cur_leaf_count_per_tree);
    hipFree(d_attribute_val_arr);
    hipFree(d_attribute_idx_arr);
    hipFree(d_confusion_matrix);

    output_file.close();

    return 0;
}
